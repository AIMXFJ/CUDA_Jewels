#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <fstream>

//funcion para generar una jewel aleatoria, como la generacion inicial.
/* Funciones para generar gemas aleatorias */
/* Iniciador de seeds */
__global__ void setup_kernel(hiprandState * state, unsigned long seed)
{
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

/* Crear jewel usando globalState */
__device__ float generate(hiprandState* globalState, int ind)
{
	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);
	globalState[ind] = localState;
	return RANDOM;
}

/* Funcion para generarJewel en CUDA */
__device__ int generarJewelCUDA(hiprandState* globalState, int ind, int dificultad)
{
	switch (dificultad) {
	case 1:
	{
		return (int)1 + generate(globalState, ind) * 4;
	}
	case 2: {
		return (int)1 + generate(globalState, ind) * 6;
	}
	case 3: {
		return (int)1 + generate(globalState, ind) * 8;
	}
	}
	return -1;
}

/* Funcion para inicializar la matriz de gemas */
__global__ void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura, hiprandState* globalState) {
	int tFila = threadIdx.y;
	int tColumna = threadIdx.x;
	if (tFila < altura)
	{
		if (tColumna < anchura)
		{
			tablero[tFila*anchura + tColumna] = generarJewelCUDA(globalState, tFila * anchura + tColumna, dificultad);
		}
	}
}

/* Funcion para imprimir el tablero en GPU */
void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j + i*anchura]);
		}
	}
	printf("\n");
}

//TODO VERTICAL
/*__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d, int dificultad, int anchura, int altura, int final) {
	//int ty = threadIdx.x + jewels_eliminadas_d[1];
	//int tx = threadIdx.y + jewels_eliminadas_d[0];
	int tx = threadIdx.y;

	printf("\ntx:%i\n",tx);

	printf("\n eliminadas jewels _Device -> ");
	for (int q = 0; q < final; q++) {
		printf("%f |", jewels_eliminadas_d[q]);
	}
	printf("\n");

	//Horizontal
	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2]) {
		int posicion_abajo = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2) + 1] - 1) * anchura;
		int posicion = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2) + 1]) * anchura;

		printf("\ntx*2:%i  tx*2+1:%i\n", tx * 2, tx * 2 + 1);
		printf("\nposiciones x:%f y:%f\n", jewels_eliminadas_d[tx * 2], jewels_eliminadas_d[tx * 2 + 1]);

		if (jewels_eliminadas_d[(tx * 2) + 1]-1 < altura) {
			tablero_d[posicion_abajo] = tablero_d[posicion];
			tablero_d[posicion] = -1;
		}
		else {
			if (jewels_eliminadas_d[(tx * 2) + 1]-1 == altura) {
				tablero_d[posicion_abajo] = -1;
			}
		}
	}//Vertical
	else {
		int posicion_arriba = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2) + 1] - 1 + final/2) * anchura;
		int posicion = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2)+1] - 1) * anchura;

		if (jewels_eliminadas_d[(tx * 2) + 1] - 1 + final / 2 < altura) {
			tablero_d[posicion] = tablero_d[posicion_arriba];
			tablero_d[posicion_arriba] = -1;
		}
		else {
			if (jewels_eliminadas_d[(tx * 2) + 1] == altura - 1)
				jewels_eliminadas_d[(tx * 2) + 1] == -1;
		}

		//float value = tablero_d[tx + (ty)*anchura];
		//tablero_d[tx + (ty - final / 2)*(anchura)] = value;
		//tablero_d[tx + (ty)*anchura] = -1;
	}

	//tablero_d[tx+(ty-1)*anchura]=tablero_d[tx + ty*anchura];
	//tablero_d[tx + ty*anchura] = -1;
}*/

//TODO: Se pisan las filas entre ellas al no ir en orden.
/*__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d,int dificultad, int anchura, int altura, int final) {
	int tx = threadIdx.x + jewels_eliminadas_d[0];
	int ty = blockIdx.y + jewels_eliminadas_d[1];
	printf("\nBidx x:%i y:%i  | thrdIdx x:%i y:%i\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	printf("\nFinal: %i\n", final);

	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2]) {
		//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
			//printf("A");
			//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
				printf("\THREAD X:%i  Y:%i\n", tx, ty);
				if (ty + 1 < altura) {
					//if ty + 1 == altura
					float value = tablero_d[tx + (ty + 1)*anchura];

					__syncthreads();

					tablero_d[tx + (ty)*(anchura)] = value;

					__syncthreads();

					tablero_d[tx + (ty + 1)*anchura] = -1;
				}
				else {
						//tablero_d[tx + ty*anchura] = -2;
				}
		//	}
	//	}
	}
	else {
		//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
			//printf("A");
			//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
				//printf("\nBUCLE X:%i  Y:%i\n", x, y);
				if (ty < altura) {
					if (ty >= jewels_eliminadas_d[final - 2]) {
						float value = tablero_d[tx + (ty)*anchura];
						tablero_d[tx + (ty - final / 2)*(anchura)] = value;
						tablero_d[tx + (ty)*anchura] = -1;
					}
					else {
						tablero_d[tx + (ty)*anchura] = -1;
					}
				}
			//}
		//}
	}

	/*if (altura >= anchura) max = altura;
	else max = anchura;
	printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);

	for (int i = 0; i < max; i++) {
		if ((tx == jewels_eliminadas_d[i]) && (ty)>(jewels_eliminadas_d[i + 1])) {
			tablero_d[tx + (ty - 1)*(anchura)] = tablero_d[tx + ty*anchura];
		}

		if (ty == altura) {
			//Generar jewel random
			hiprandState state;

			hiprand_init((unsigned long long)clock(), i, 0, &state);

			tablero_d[tx + ty*anchura] = hiprand_uniform(&state);
		}
		i++;
	}*
}*/

//Elimina las jewels recibidas, bajas las filas para rellenas, y genera arriba del todo jewels nuevas. TODO
/*void eliminarJewels(float* tablero, float* jewels_eliminadas,int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *aux;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	aux = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < 2*max; i++) {
		aux[i]=jewels_eliminadas[i];
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
	//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
	int final = 0;

	for (int i = 0; i < max * 2; i++) {
		printf("\ni:%i valor:%f\n", i, jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max * 2;

	//Configuracion de ejecucion, 1 bloque por fila con tantos hilos como columnas
	//dim3 dimBlock(altura-jewels_eliminadas[1]-1,1);
	//dim3 dimGrid(1,jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);

	//n� de bloques
	dim3 dimGrid(1,1);

	printf("\nfinal: %i\n",final);
	printf("\n");
	for (int w = 0; w < final; w++) {
		printf("%f |",aux[w]);
	}
	printf("\n");

	if(aux[0]>=0)
	for (int z = 1; z <= altura-aux[1]-1; z++) {
		printf("\nantes buc k <= %f\n", aux[final - 2] - aux[0]);
		for (int k = 0; k < final; k+=2) {
			jewels_eliminadas[k] = aux[k];
			jewels_eliminadas[k + 1] = aux[k + 1] + z;
			printf("\na�adido a eliminadas x:%f y:%f\n",aux[k],aux[k+1]+z);
		}

		printf("\n eliminadas jewels -> ");
		for (int q = 0; q < final; q++) {
			printf("%f |", jewels_eliminadas[q]);
		}
		printf("\n");

		//Inicio del calculo, misma funcion de analisis en manual y automatico
		hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

		if (jewels_eliminadas[1] == jewels_eliminadas[3]) {
			dim3 dimBlock(1, jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);
			eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
		}
		else {
			dim3 dimBlock(altura - jewels_eliminadas[1] + 1, 1);
			eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
		}
		printf("\nLLAMADA\n");

		//Transfiere las jewels a eliminar de la GPU al host
		hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	}

	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}*/

__global__ void eliminarJewelsKernel(float* tablero_d, float* tablero_aux_d, float* jewels_eliminadas_d, int dificultad, int anchura, int altura, int final) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	//printf("\nBidx x:%i y:%i  | thrdIdx x:%i y:%i\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//printf("\nFinal: %i\n", final);

	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2] && tx >= jewels_eliminadas_d[0] && tx <= jewels_eliminadas_d[final - 2] && ty >= jewels_eliminadas_d[1]) {
		//printf("\THREAD X:%i  Y:%i\n", tx, ty);
		if (ty + 1 < altura) {
			float value = tablero_aux_d[tx + (ty + 1)*anchura];

			//printf("\nvalue: %f\n",value);

			tablero_d[tx + (ty)*(anchura)] = value;

			//tablero_d[tx + (ty + 1)*anchura] = -1;
		}
		else {
			//printf("\nFin\n");
			tablero_d[tx + ty*anchura] = -1;
		}
	}
	else {
		if (ty < altura && tx == jewels_eliminadas_d[0] && ty > jewels_eliminadas_d[1]) {
			float value = tablero_aux_d[tx + (ty)*anchura];
			tablero_d[tx + (ty - final / 2)*(anchura)] = value;
			//tablero_d[tx + (ty)*anchura] = -1;
		}
		if (ty >= altura - final / 2 && ty < altura && tx == jewels_eliminadas_d[0]) {
			tablero_d[tx + (ty)*anchura] = -1;
		}
	}
}

void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *tablero_aux_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&tablero_aux_d, size);
	hipMemcpy(tablero_aux_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	dim3 dimGrid(1, 1);
	dim3 dimBlock(anchura, altura);
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

	int final = 0;

	for (int i = 0; i < max * 2; i++) {
		printf("\ni:%i valor:%f\n", i, jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max * 2;

	eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, tablero_aux_d, jewels_eliminadas_d, dificultad, anchura, altura, final);

	hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	for (int k = 0; k < size; k++) {
		if (tablero[k] == -1) {
			srand(time(NULL));
			switch (dificultad) {
			case 1: {
				int randJewel = rand() % 4 + 1;
				tablero[k] = randJewel;
				break;
			}
			case 2: {
				int randJewel = rand() % 6 + 1;
				tablero[k] = randJewel;
				break;
			}
			case 3: {
				int randJewel = rand() % 8 + 1;
				tablero[k] = randJewel;
				break;
			}
			};
		}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
	hipFree(tablero_aux_d);
}

/*void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *aux;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	aux = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < 2 * max; i++) {
		aux[i] = jewels_eliminadas[i];
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
	//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
	int final = 0;

	for (int i = 0; i < max * 2; i++) {
		printf("\ni:%i valor:%f\n", i, jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max * 2;

	//Configuracion de ejecucion, 1 bloque por fila con tantos hilos como columnas
	//dim3 dimBlock(altura-jewels_eliminadas[1]-1,1);
	//dim3 dimGrid(1,jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);

	//n� de bloques
	dim3 dimGrid(1, 1);

	printf("\nfinal: %i\n", final);
	printf("\n");
	for (int w = 0; w < final; w++) {
		printf("%f |", aux[w]);
	}
	printf("\n");

	if (aux[0] >= 0)
		for (int z = 1; z <= altura - aux[1] - 1; z++) {
			printf("\nantes buc k <= %f\n", aux[final - 2] - aux[0]);
			for (int k = 0; k < final; k += 2) {
				jewels_eliminadas[k] = aux[k];
				jewels_eliminadas[k + 1] = aux[k + 1] + z;
				printf("\na�adido a eliminadas x:%f y:%f\n", aux[k], aux[k + 1] + z);
			}

			printf("\n eliminadas jewels -> ");
			for (int q = 0; q < final; q++) {
				printf("%f |", jewels_eliminadas[q]);
			}
			printf("\n");

			//Inicio del calculo, misma funcion de analisis en manual y automatico
			hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

			if (jewels_eliminadas[1] == jewels_eliminadas[3]) {
				dim3 dimBlock(1, jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);
				eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
			}
			else {
				dim3 dimBlock(altura - jewels_eliminadas[1] + 1, 1);
				eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
			}
			printf("\nLLAMADA\n");

			//Transfiere las jewels a eliminar de la GPU al host
			hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

		}

	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}*/

__global__ void analisisTableroAutomaticoKernel(float *tablero_d, float *aux_d, int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int jewels_posibles_der = 0;

	//Si tiene por la derecha
	if ((tx + 2) < anchura) {
		if (((tx + 2) + ty*anchura <= altura*anchura) && tablero_d[tx + 2 + ty*anchura] == tablero_d[tx + ty*anchura]) {
			int i = 2;
			while ((tx + i + ty*anchura <= altura*anchura) && tablero_d[tx + i + ty*anchura] == tablero_d[tx + ty*anchura]) {
				jewels_posibles_der++;
				i++;
			}

			aux_d[tx + ty*anchura] = jewels_posibles_der + 1;
		}
		else {
			aux_d[tx + ty*anchura] = 1;
		}
	}
	else {
		aux_d[tx + ty*anchura] = 1;
	}
}

void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	int jewels_posibles_izq = 0;
	int jewels_posibles_der = 0;
	//printf("\nHORIZONTAL\n");
	//Si tiene por la izquierda
	if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x - i + y*anchura >= 0) && (x - i >= 0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_izq++;
			i++;
		}
	}

	//Si tiene por la derecha
	if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x + i + y*anchura <= size) && (x + i < anchura) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_der++;
			i++;
		}
	}

	//Se pueden eliminar horizontalmente
	if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
		int salto = 0;

		//printf("\nIZQ:%i   DER:%i\n",jewels_posibles_izq,jewels_posibles_der);

		for (int j = jewels_posibles_izq; j >= (1); j--) {
			jewels_eliminadas[salto] = x - j;
			jewels_eliminadas[salto + 1] = y;
			salto += 2;
		}

		jewels_eliminadas[jewels_posibles_izq * 2] = x;
		jewels_eliminadas[jewels_posibles_izq * 2 + 1] = y;

		salto = 2;
		for (int k = 1; k <= jewels_posibles_der; k++) {
			jewels_eliminadas[salto + jewels_posibles_izq * 2] = x + k;
			jewels_eliminadas[salto + jewels_posibles_izq * 2 + 1] = y;
			salto += 2;
		}
	}
	else {	//Analizamos la vertical
		int jewels_posibles_arrib = 0;
		int jewels_posibles_abaj = 0;

		//printf("\nVERTICAL\n");
		//Si tiene por abajo
		if ((x + (y - 1)*anchura >= 0) && tablero[x + (y - 1)*anchura] == tablero[x + y*anchura]) {
			printf("\nABAJO\n");
			int i = 1;
			while ((x + (y - i)*anchura >= 0) && tablero[x + (y - i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_abaj++;
				//printf("\nTIENE ABAJO\n");
				i++;
			}
		}

		//Si tiene por arriba
		if ((x + 1 + y*anchura <= size) && tablero[x + (y + 1)*anchura] == tablero[x + y*anchura]) {
			//printf("\nARRIBA\n");
			int i = 1;
			while ((x + (y + i)*anchura <= size) && tablero[x + (y + i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_arrib++;
				//printf("\nTIENE ARRIBA\n");
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {
			//printf("\nSE PUEDE\n");

			int salto = 0;
			for (int j = jewels_posibles_abaj; j >= (1); j--) {
				jewels_eliminadas[salto] = x;
				jewels_eliminadas[salto + 1] = y - j;
				salto += 2;
			}

			jewels_eliminadas[jewels_posibles_abaj * 2] = x;
			jewels_eliminadas[jewels_posibles_abaj * 2 + 1] = y;

			salto = 2;
			for (int k = 1; k <= jewels_posibles_arrib; k++) {
				jewels_eliminadas[salto + jewels_posibles_abaj * 2] = x;
				jewels_eliminadas[salto + jewels_posibles_abaj * 2 + 1] = y + k;
				salto += 2;
			}
		}
	}

	for (int q = 0; q < 2 * max; q++) {
		if (q % 2 != 0) {
			printf(" y:%f\n", jewels_eliminadas[q]);
		}
		else {
			printf("| x:%f\n", jewels_eliminadas[q]);
		}
	}
	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura, int seleccion, int dificultad) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion)
	{
	case 1: //Arriba
	{
		jewel2_y += 1;
		break;
	}
	case 2: //Abajo
	{
		jewel2_y -= 1;
		break;
	}
	case 3: //Izquierda
	{
		jewel2_x -= 1;
		break;
	}
	case 4: //Derecha
	{
		jewel2_x += 1;
		break;
	}
	}
	int aux1;

	aux1 = tablero[jewel2_x + jewel2_y*anchura];

	tablero[jewel2_x + jewel2_y*anchura] = tablero[jewel1_x + jewel1_y*anchura];
	tablero[jewel1_x + jewel1_y*anchura] = aux1;

	analisisTableroManual(dificultad, tablero, anchura, altura, jewel2_x, jewel2_y);
}

//CUDA CPU Function. Analiza la mejor opcion y la ejecuta
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *aux_d;
	float *aux;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));
	aux = (float*)malloc(size);

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	for (int p = 0; p < size; p++) {
		aux[p] = 1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&aux_d, size);
	hipMemcpy(aux_d, aux, size, hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	analisisTableroAutomaticoKernel <<<dimGrid,dimBlock>>> (tablero_d, aux_d, dificultad, anchura, altura);
	if (hipSuccess != hipGetLastError())
		printf("\nCUDA Error!\n");

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(aux, aux_d, size, hipMemcpyDeviceToHost);

	printTablero(aux, anchura, altura);

	int x_mejor = 0;
	int y_mejor = 0;
	int valor_mejor = 0;

	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			if (aux[x+y*anchura] > valor_mejor) {
				valor_mejor = aux[x+y*anchura];
				x_mejor = x;
				y_mejor = y;
			}
		}
	}

	if (valor_mejor >= 3) {
		intercambiarPosiciones(tablero, x_mejor, y_mejor, 4, anchura, altura, 1, dificultad);
	}
}

/* Funcion que carga del archivo la anchura, altura y dificultad del tablero */
bool precargar(int& anchura, int& altura, int& dificultad, char* fichero)
{
	std::ifstream fCarga(fichero);
	char tam[4];
	if (!fCarga.is_open())
	{
		std::cout << "ERROR: no existe un archivo guardado." << std::endl;
		return false;
	}

	fCarga.getline(tam, 4);

	anchura = (int)tam[0] - 48;
	altura = (int)tam[1] - 48;
	dificultad = (int)tam[2] - 48;

	fCarga.close();
	return true;
}

/* Funcion que carga el tablero guardado previamente */
void cargar(int anchura, int altura, float*  tablero, char* fichero)
{
	char* array = (char*)malloc(anchura*altura + 1 + 3);
	std::ifstream fCarga(fichero);
	fCarga.getline(array, (anchura*altura + 1 + 3));
	for (int i = 0; i < anchura*altura; i++)
	{
		tablero[i] = array[i + 3] - 48;
	}
	free(array);
	fCarga.close();
}

/* Funcion que guarda el tablero */
void guardado(float* tablero, int anchura, int altura, int dificultad, char* fichero)
{
	//Sistema de guardado
	std::ofstream ficheroGuardado;
	ficheroGuardado.open(fichero);
	ficheroGuardado.clear();
	/* Almacenar anchura y altura*/
	ficheroGuardado << anchura;
	ficheroGuardado << altura;
	ficheroGuardado << dificultad;
	/* Almacenar Resto */
	for (int index = 0; index < anchura*altura; index++)
	{
		ficheroGuardado << tablero[index];
	}
	ficheroGuardado.close();
}

/* Funcion que elimina una fila */
__global__ void bombaFila(float* tablero, int anchura, int altura, int dificultad, int fila, hiprandState* globalState) {

	int tFila = threadIdx.y;
	int tColumna = threadIdx.x;

	if ((tFila + fila) < altura)
	{
		if (tColumna < anchura)
		{
			if ((tFila + fila + 1) == altura)
			{
				tablero[(tFila + fila)*anchura + tColumna] = generarJewelCUDA(globalState, (tFila * 3 + tColumna), dificultad);
			}
			else {
				tablero[(tFila + fila)*anchura + tColumna] = tablero[(tFila + fila + 1)*anchura + tColumna];

			}
		}
	}
}

/* Funcion que elimina una columna */
__global__ void bombaColumna(float* tablero, int anchura, int altura, int dificultad, int columna, hiprandState* globalState) {

	int tFila = threadIdx.y;
	int tColumna = threadIdx.x;

	if (tFila < altura)
	{
		if ((tColumna + columna) < anchura)
		{
			if ((tColumna + columna + 1) == anchura)
			{
				tablero[(tFila*anchura) + (tColumna + columna)] = generarJewelCUDA(globalState, (tFila * 3 + tColumna), dificultad);
			}
			else {
				tablero[(tFila*anchura) + (tColumna + columna)] = tablero[(tFila*anchura) + (tColumna + columna + 1)];
			}
		}
	}
}

__global__ void bombaRotarGPU1(float* tablero, int anchura, int altura)
{
	int tFila = threadIdx.y;
	int tColumna = threadIdx.x;
	int fila = -1, columna = -1;
	float aux[9];

	if (tFila < altura)	{
		if (tColumna < anchura)	{
			if ((tFila - 1) < 0 || (tFila + 1) >= altura || (tColumna - 1) < 0 || (tColumna + 1) >= anchura) {}
			else {
				if (tFila*anchura + tColumna % 4 == 1) {
					fila = tFila;
					columna = tColumna;
				}
				if (fila != -1 && columna != -1)
				{
					aux[tFila * 3 + tColumna] = tablero[((fila + 1) - tFila) + ((columna + 1) - tColumna)*altura];
					printf("%f", aux[tFila * 3 + tColumna]);
					tablero[((fila + 1) - tFila)*anchura + ((columna - 1) + tColumna)] = aux[tFila * 3 + tColumna];
				}
			}
		}
	}
}

__global__ void bombaRotarGPU(float* tablero, int anchura, int altura, int fila, int columna)
{
	float aux[9];
	int tFila = threadIdx.y;
	int tColumna = threadIdx.x;

	if (tFila < 3)
	{
		if (tColumna < 3)
		{
			aux[tFila + tColumna * 3] = tablero[((fila + 1) - tFila) *anchura + ((columna + 1) - tColumna)];
			printf("%i-%i ", tFila + tColumna * 3, aux[tFila+tColumna*3]);
			tablero[((fila + 1) - tFila)*anchura + ((columna - 1) + tColumna)] = aux[tFila * 3 + tColumna];
			printf("%i_%i ", ((fila + 1) - tFila)*anchura + ((columna - 1) + tColumna), tFila * 3 + tColumna);
		}
	}
}

__global__ void bombaRotar(float* tablero_d, int anchura, int altura)
{
	int tFila = threadIdx.y;
	int tColumna = threadIdx.x;
	if (tFila < altura && tColumna < anchura) {
		if ((tFila - 1) < 0 || (tFila + 1) >= altura || (tColumna - 1) < 0 || (tColumna + 1) >= anchura)
		{
			/* Se entra cuando no se puede rotar */

		}
		else
		{
			if (tFila % 3 == 1 && tColumna % 3 == 1)
			{
				dim3 dimBlock(3, 3);
				dim3 dimGrid(1, 1);
				
				bombaRotarGPU << <dimGrid, dimBlock >> >(tablero_d, anchura, altura, tFila, tColumna);
				//__syncthreads();
			}
		}
	}
}

int main(int argc, char** argv) {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura, altura, dificultad, size, seleccion;
	int jewel1_x, jewel1_y, accion;
	char modo, ficheroGuardado[9] = "save.txt";;
	bool automatico = true;
	bool encontrado = false;
	bool jugando = true;
	float* tablero;
	float* tablero_d;

	hiprandState* devStates;

	/* Valores por argumento/
	modo = argv[1][1];
	dificultad = atoi(argv[2]);
	anchura = atoi(argv[3]);
	altura = atoi(argv[4]);*/

	modo = 'a';
	dificultad = 3;
	anchura = 6;
	altura = 3;
	size = anchura*altura;

	/* Establecer modo de juego */
	switch (modo) {
	case 'a': {seleccion = 1; break; }
	case 'm': {seleccion = 2; break; }
	default: printf("Valor no valido.\n"); return -1;
	}

	/* Inicializacion random en CUDA */
	hipMalloc(&devStates, size * sizeof(hiprandState));
	/* Creacion de las Seeds */
	setup_kernel << < 1, size >> > (devStates, unsigned(time(NULL)));

	/* Reservar memoria para tablero y tablero_d */
	tablero = (float*)malloc(size * sizeof(float));
	hipMalloc((void**)&tablero_d, size * sizeof(float));

	/* Inicializacion de la Matriz en CUDA*/
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);
	generacionInicialRandomJewels << < dimGrid, dimBlock >> > (tablero_d, dificultad, anchura, altura, devStates);
	hipMemcpy(tablero, tablero_d, size * sizeof(float), hipMemcpyDeviceToHost);

	//Bucle principal del juego
	while (jugando) {

		printTablero(tablero, anchura, altura);

		jewel1_x = 0;
		jewel1_y = 0;
		accion = 0;

		std::cout << "Acci�n a realizar:\n";
		std::cout << "(1) Intercambiar Jewels\n";
		std::cout << "(2) Guardar partida\n";
		std::cout << "(3) Cargar partida\n";
		std::cout << "(9) Usar una Bomba\n";
		std::cout << "(0) Exit\n";
		std::cout << "Elija accion: ";

		std::cin >> accion;

		switch (accion) {
		/* Salir*/
		case 0: {
			goto Salir;
			break;
		}
		/* Intercambio */
		case 1: {

			std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
			std::cout << "X: ";
			std::cin >> jewel1_x;
			std::cout << "Y: ";
			std::cin >> jewel1_y;

			if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
				printf("Posicion erronea.\n");
				continue;
			}

			int direccion = 0;
			std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
			std::cin >> direccion;

			if (direccion > 4 && direccion > 1) {
				printf("Direccion erronea.\n");
				continue;
			}
			else {
				switch (direccion)
				{
				case 1: //Arriba
				{
					if (jewel1_y == altura)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 2: //Abajo
				{
					if (jewel1_y == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 3: //Izquierda
				{
					if (jewel1_x == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 4: //Derecha
				{
					if (jewel1_x == anchura - 1)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				}

				intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura, seleccion, dificultad);

				if (seleccion == 1)
					analisisTableroAutomatico(dificultad, tablero, anchura, altura);
			}

			break;
		}
		/* Guardar tablero */
		case 2: {
			guardado(tablero, anchura, altura, dificultad, ficheroGuardado);
			std::cout << "Guardado correcto.\n";
			break;
		}
		/* Cargar tablero */
		case 3: {
			/* Precarga de tablero */
			encontrado = precargar(anchura, altura, dificultad, ficheroGuardado);

			if (encontrado)
			{
				/* Cargar tablero */
				cargar(anchura, altura, tablero, ficheroGuardado);
				std::cout << "Se ha cargado el Tablero: \n";
			}
			else {
				std::cout << "No existe ninguna partida guardada.\n";
			}
			break;

		}
		/* Usar Bombas */
		case 9: {
			int bomba = 0, fila = 0, columna = 0;
			std::cout << "Elija una bomba:";

			/* CUDA */
			dim3 blockDim(anchura, altura);
			dim3 blockGrid(1, 1);
			hipMemcpy(tablero_d, tablero, size * sizeof(float), hipMemcpyHostToDevice);

			/* Bombas por tipo de dificultad */
			switch (dificultad) {
			/* Dificultad 1 */
			case 1: {
				std::cout << "\n(1) Bomba de fila ";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				std::cout << "Fila: ";
				std::cin >> fila;
				bombaFila << < dimGrid, dimBlock >> > (tablero_d, anchura, altura, dificultad, fila, devStates);
				break;
			}
			/* Dificultad 2 */
			case 2: {
				std::cout << "\n(1) Bomba de fila";
				std::cout << "\n(2) Bomba de columna";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "Fila: ";
					std::cin >> fila;
					bombaFila << < dimGrid, dimBlock >> > (tablero_d, anchura, altura, dificultad, fila, devStates);
					break;
				}
				case 2:
				{
					std::cout << "Columna: ";
					std::cin >> columna;
					bombaColumna << <dimGrid, dimBlock >> > (tablero_d, anchura, altura, dificultad, columna, devStates);
					break;
				}
				}
				break;
			}
			/* Dificultad 3 */
			case 3: {
				std::cout << "\n(1) Bomba de fila";
				std::cout << "\n(2) Bomba de columna";
				std::cout << "\n(3) Bomba de rotacion 3x3";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "Fila: ";
					std::cin >> fila;
					bombaFila << < dimGrid, dimBlock >> > (tablero_d, anchura, altura, dificultad, fila, devStates);
					break;
				}
				case 2:
				{
					std::cout << "Columna: ";
					std::cin >> columna;
					bombaColumna << <dimGrid, dimBlock >> > (tablero_d, anchura, altura, dificultad, columna, devStates);
					break;
				}
				case 3: 
				{
					for (int fila = 1; fila < anchura; fila += 3)
					{
						for (int columna = 1; columna < altura; columna += 3)
						{
							if ((fila - 1) < 0 || (fila + 1) >= altura || (columna - 1) < 0 || (columna + 1) >= anchura)
							{
								/* Se entra cuando no se puede rotar */
							}
							else
							{
								dim3 blockGrid(anchura/3, altura/3);
								bombaRotar << < dimGrid, dimBlock >> > (tablero_d, anchura, altura);
							}
						}
					}
					break;
				}
				}

				break;
			}
			}
			/* Actualizar Tablero */
			hipMemcpy(tablero, tablero_d, size * sizeof(float), hipMemcpyDeviceToHost);
			break;
		}
		}
	}

Salir:
	free(tablero);
	hipFree(tablero_d);
	hipFree(devStates);
	return 0;
}