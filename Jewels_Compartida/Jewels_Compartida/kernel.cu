#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <fstream>

//Analiza las propiedades de la tarjeta grafica para devolver el tama�o adecuado de tile, tambien trata el tama�o del tablero
int obtenerTileWidth(int anchura, int altura) {
	int min_medida = 0;

	if (anchura > altura) min_medida = anchura;
	else min_medida = altura;

	hipDeviceProp_t propiedades;

	int max_threads = propiedades.maxThreadsPerBlock;

	if (anchura == altura) {	//Si la matriz es cuadrada, para no tener 1 solo bloque
		if (min_medida / 32 > 1 && max_threads == 1024) { //Solo si tiene 1024 hilos por bloque podra ser de 32x32
			return 32;
		}
		if (min_medida / 16 > 1) {
			return 16;
		}
		else if (min_medida / 8 > 1) {
			return 8;
		}
		else if (min_medida / 4 > 1) {
			return 4;
		}
		else if (min_medida / 2 > 1) {
			return 2;
		}
	}
	else {	//si la matriz no es cuadrada
		if (min_medida / 32 >= 1 && max_threads == 1024) {
			return 32;
		}
		if (min_medida / 16 >= 1) {
			return 16;
		}
		else if (min_medida / 8 >= 1) {
			return 8;
		}
		else if (min_medida / 4 >= 1) {
			return 4;
		}
		else if (min_medida / 2 >= 1) {
			return 2;
		}
	}
}

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
	return -1;
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	srand(time(NULL));
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}
void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j + i*anchura]);
		}
	}
	printf("\n");
}

/*Recibe las coordenadas de las jewels a eliminar y mueve las filas que tiene que bajar a partir de ellas, emplea
una copia del tablero para evitar race conditions*/
__global__ void eliminarJewelsKernel(float* tablero_d, float* tablero_aux_d, float* jewels_eliminadas_d, int dificultad, int anchura, int altura, int final, int TILE_WIDTH) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;

	//Posicion real dentro del tablero
	tx += block_x * TILE_WIDTH;
	ty += block_y * TILE_WIDTH;

	//Array dinamico en memoria compartida, velocidad de accesoo mucho mayor que con global
	extern __shared__ float tablero_aux_shared[];

	float* tablero_shared = tablero_aux_shared;

	//Entre todos los hilos, los cuales ponen su posicion en el auxiliar compartido, rellenan por completo el auxiliar
	tablero_shared[tx + ty*anchura] = tablero_aux_d[tx + ty*anchura];

	//Esperan a que todos los hilos pongan su jewel, creando un tablero auxiliar completo en compartida.
	__syncthreads();

	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2] && tx >= jewels_eliminadas_d[0] && tx <= jewels_eliminadas_d[final - 2] && ty >= jewels_eliminadas_d[1]) {
		if (ty + 1 < altura) {
			float value = tablero_shared[tx + (ty + 1)*anchura];

			tablero_d[tx + (ty)*(anchura)] = value;
		}
		else {
			tablero_d[tx + ty*anchura] = -1;
		}
	}
	else {
		if (ty < altura && tx == jewels_eliminadas_d[0] && ty > jewels_eliminadas_d[1]) {
			float value = tablero_shared[tx + (ty)*anchura];
			tablero_d[tx + (ty - final / 2)*(anchura)] = value;
		}
		if (ty >= altura - final / 2 && ty < altura && tx == jewels_eliminadas_d[0]) {
			tablero_d[tx + (ty)*anchura] = -1;
		}
	}
}

/*Funcion que prepara y llama el kernel con su mismo nombre, genera todos los datos necesarios*/
void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura,int TILE_WIDTH) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *tablero_aux_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	//Para saber que medida es la m�s grande, ya que no se pueden eliminar m�s jewels seguidas que esa medida
	if (altura >= anchura) max = altura;
	else max = anchura;

	//Tablero a GPU y la copia del tablero
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&tablero_aux_d, size);
	hipMemcpy(tablero_aux_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU. 2*max ya que cada posicion son dos coordenadas, x e y
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

	int final = 0;

	//Calcula cual es el ultimo valor escrito de las jewels a eliminar, ya que puede haber posiciones no escritas
	for (int i = 0; i < max * 2; i++) {
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	//En caso de que este completamente escrito
	if (final == 0) final = max * 2;

	//Cantidad de bloques de ancho de medida TILE_WIDTH
	int anch = ceil(anchura / TILE_WIDTH);

	//Cantidad de bloques de alto con medida TILE_WIDTH
	int alt = ceil(altura / TILE_WIDTH);

	//Configuracion de ejecucion
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	dim3 dimGrid(alt, anch);

	eliminarJewelsKernel << <dimGrid, dimBlock, size>> > (tablero_d, tablero_aux_d, jewels_eliminadas_d, dificultad, anchura, altura, final,TILE_WIDTH);

	//Se recupera el tablero actualizado
	hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	//Se analiza y se a�aden jewels nuevas en las posiciones vacias (-1)
	for (int k = 0; k < size; k++) {
		if (tablero[k] == -1) {
			srand(time(NULL));
			switch (dificultad) {
			case 1: {
				int randJewel = rand() % 4 + 1;
				tablero[k] = randJewel;
				break;
			}
			case 2: {
				int randJewel = rand() % 6 + 1;
				tablero[k] = randJewel;
				break;
			}
			case 3: {
				int randJewel = rand() % 8 + 1;
				tablero[k] = randJewel;
				break;
			}
			};
		}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
	hipFree(tablero_aux_d);
}

//Analiza el movimiento manual, usando las coordenadas de la nueva posicion de la jewel seleccionada
void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y, int TILE_WIDTH) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	//Se inicializa a -1 �ra saber hasta que punto se escribe
	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	int jewels_posibles_izq = 0;
	int jewels_posibles_der = 0;

	//Si tiene por la izquierda
	if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x - i + y*anchura >= 0) && (x - i >= 0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_izq++;
			i++;
		}
	}

	//Si tiene por la derecha
	if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x + i + y*anchura <= size) && (x + i < anchura) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_der++;
			i++;
		}
	}

	//Se pueden eliminar horizontalmente, las coloca en orden para facilitar su eliminacion
	if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
		int salto = 0;

		for (int j = jewels_posibles_izq; j >= (1); j--) {
			jewels_eliminadas[salto] = x - j;
			jewels_eliminadas[salto + 1] = y;
			salto += 2;
		}

		jewels_eliminadas[jewels_posibles_izq * 2] = x;
		jewels_eliminadas[jewels_posibles_izq * 2 + 1] = y;

		salto = 2;
		for (int k = 1; k <= jewels_posibles_der; k++) {
			jewels_eliminadas[salto + jewels_posibles_izq * 2] = x + k;
			jewels_eliminadas[salto + jewels_posibles_izq * 2 + 1] = y;
			salto += 2;
		}
	}
	else {	//Analizamos la vertical
		int jewels_posibles_arrib = 0;
		int jewels_posibles_abaj = 0;

		//Si tiene por abajo
		if ((x + (y - 1)*anchura >= 0) && tablero[x + (y - 1)*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x + (y - i)*anchura >= 0) && tablero[x + (y - i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_abaj++;
				i++;
			}
		}

		//Si tiene por arriba
		if ((x + 1 + y*anchura <= size) && tablero[x + (y + 1)*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x + (y + i)*anchura <= size) && tablero[x + (y + i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_arrib++;
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {

			int salto = 0;
			for (int j = jewels_posibles_abaj; j >= (1); j--) {
				jewels_eliminadas[salto] = x;
				jewels_eliminadas[salto + 1] = y - j;
				salto += 2;
			}

			jewels_eliminadas[jewels_posibles_abaj * 2] = x;
			jewels_eliminadas[jewels_posibles_abaj * 2 + 1] = y;

			salto = 2;
			for (int k = 1; k <= jewels_posibles_arrib; k++) {
				jewels_eliminadas[salto + jewels_posibles_abaj * 2] = x;
				jewels_eliminadas[salto + jewels_posibles_abaj * 2 + 1] = y + k;
				salto += 2;
			}
		}
	}

	//Las elimina
	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura, TILE_WIDTH);
}

//Intercambia la jewel seleccionadas con la jewel en la direcci�n indicada
void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura, int seleccion, int dificultad, int TILE_WIDTH) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion)
	{
	case 1: //Arriba
	{
		jewel2_y += 1;
		break;
	}
	case 2: //Abajo
	{
		jewel2_y -= 1;
		break;
	}
	case 3: //Izquierda
	{
		jewel2_x -= 1;
		break;
	}
	case 4: //Derecha
	{
		jewel2_x += 1;
		break;
	}
	}
	int aux1;

	aux1 = tablero[jewel2_x + jewel2_y*anchura];

	tablero[jewel2_x + jewel2_y*anchura] = tablero[jewel1_x + jewel1_y*anchura];
	tablero[jewel1_x + jewel1_y*anchura] = aux1;

	//Analiza el movimiento para ver si se pueden eliminar jewels
	analisisTableroManual(dificultad, tablero, anchura, altura, jewel2_x, jewel2_y,TILE_WIDTH);
}

/*Escribe en un tablero auxiliar la cantidad de jewels que se eliminarian moviendo una jewel (x,y) hacia la derecha
paralelizable ya que todos los hilos (cada hilo 1 jewel) tienen que expandirse hacia la derecha para ver hasta donde llegarian a eliminar*/
__global__ void analisisTableroAutomaticoKernel(float *tablero_d, float *aux_d, int dificultad, int anchura, int altura, int TILE_WIDTH) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;

	//Posicion real dentro del tablero
	tx += block_x * TILE_WIDTH;
	ty += block_y * TILE_WIDTH;

	//Array dinamico en memoria compartida, velocidad de accesoo mucho mayor que con global
	extern __shared__ float tablero_aux_shared[];

	float* tablero_shared = tablero_aux_shared;

	//Entre todos los hilos, rellenan por completo el auxiliar en memoria compartida
	tablero_shared[tx + ty*anchura] = aux_d[tx + ty*anchura];

	//Esperan a que todos los hilos copien el valor, creando un tablero auxiliar completo en compartida.
	__syncthreads();

	int jewels_posibles_der = 0;

	//Si tiene por la derecha
	if ((tx + 2) < anchura) {
		if (((tx + 2) + ty*anchura <= altura*anchura) && tablero_d[tx + 2 + ty*anchura] == tablero_d[tx + ty*anchura]) {
			int i = 2;
			//Se expande
			while ((tx + i + ty*anchura <= altura*anchura) && tablero_d[tx + i + ty*anchura] == tablero_d[tx + ty*anchura]) {
				jewels_posibles_der++;
				i++;
			}

			tablero_shared[tx + ty*anchura] = jewels_posibles_der + 1;
		}
		else {
			tablero_shared[tx + ty*anchura] = 1;
		}
	}
	else {
		tablero_shared[tx + ty*anchura] = 1;
	}

	//Se esperan a que todos hayan calculado para actualizar la matriz a devolver
	__syncthreads();

	aux_d[tx + ty*anchura] = tablero_shared[tx + ty*anchura];
}


//Analiza la mejor opcion y la ejecuta en funcion de lo que devuelve el kernel
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura, int TILE_WIDTH) {
	float *tablero_d;
	float *aux_d;
	float *aux;
	float *jewels_eliminadas_d;
	//Tama�o del tablero para asignar memoria
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan max jewels, 2 coordenadas por jewel = 2 * max posiciones
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));
	aux = (float*)malloc(size);

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Solo se cuenta la jewel que se escoge, sigue siendo menor que 3
	for (int p = 0; p < size; p++) {
		aux[p] = 1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);
	//Auxiliar de conteo a GPU
	hipMalloc((void**)&aux_d, size);
	hipMemcpy(aux_d, aux, size, hipMemcpyHostToDevice);

	//Cantidad de bloques de ancho de medida TILE_WIDTH
	int anch = ceil(anchura / TILE_WIDTH);

	//Cantidad de bloques de alto con medida TILE_WIDTH
	int alt = ceil(altura / TILE_WIDTH);

	//Configuracion de ejecucion
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
	dim3 dimGrid(alt, anch);

	//Inicio del kernel
	analisisTableroAutomaticoKernel <<<dimGrid,dimBlock,size>>> (tablero_d, aux_d, dificultad, anchura, altura, TILE_WIDTH);

	//Transfiere el resultado de la GPU al host
	hipMemcpy(aux, aux_d, size, hipMemcpyDeviceToHost);

	int x_mejor = 0;
	int y_mejor = 0;
	int valor_mejor = 0;

	//Se busca el movimiento con el mayor numero de jewels eliminadas
	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			if (aux[x+y*anchura] > valor_mejor) {
				valor_mejor = aux[x+y*anchura];
				x_mejor = x;
				y_mejor = y;
			}
		}
	}

	//Si se pueden eliminar se ejecuta el movimiento, con lo que ello conlleva
	if (valor_mejor >= 3) {
		intercambiarPosiciones(tablero, x_mejor, y_mejor, 4, anchura, altura, 1, dificultad,TILE_WIDTH);
	}
}

bool precargar(int& anchura, int& altura, int& dificultad, char* fichero)
{
	std::ifstream fCarga(fichero);
	char tam[4];
	if (!fCarga.is_open())
	{
		std::cout << "ERROR: no existe un archivo guardado." << std::endl;
		return false;
	}

	fCarga.getline(tam, 4);

	anchura = (int)tam[0] - 48;
	altura = (int)tam[1] - 48;
	dificultad = (int)tam[2] - 48;

	fCarga.close();
	return true;
}
void cargar(int anchura, int altura, float*  tablero, char* fichero)
{
	char* array = (char*)malloc(anchura*altura + 1 + 3);
	std::ifstream fCarga(fichero);
	fCarga.getline(array, (anchura*altura + 1 + 3));
	for (int i = 0; i < anchura*altura; i++)
	{
		tablero[i] = array[i + 3] - 48;
	}
	free(array);
	fCarga.close();
}

void guardado(float* tablero, int anchura, int altura, int dificultad, char* fichero)
{
	//Sistema de guardado
	std::ofstream ficheroGuardado;
	ficheroGuardado.open(fichero);
	ficheroGuardado.clear();
	/* Almacenar anchura y altura*/
	ficheroGuardado << anchura;
	ficheroGuardado << altura;
	ficheroGuardado << dificultad;
	/* Almacenar Resto */
	for (int index = 0; index < anchura*altura; index++)
	{
		ficheroGuardado << tablero[index];
	}
	ficheroGuardado.close();
}
void bombaFila(float* tablero, int anchura, int altura, int dificultad, int fila) {

	for (int iFila = 0; (iFila + fila) < altura; iFila++)
	{
		for (int iColm = 0; iColm < anchura; iColm++)
		{
			if ((iFila + fila + 1) < altura)
			{
				tablero[(iFila + fila)*anchura + iColm] = tablero[(iFila + fila + 1)*altura + iColm];
			}
			else {
				tablero[(iFila + fila)*anchura + iColm] = generarJewel(dificultad);
			}
		}
	}
}

void bombaColumna(float* tablero, int anchura, int altura, int dificultad, int columna) {

	for (int iFila = 0; iFila < altura; iFila++)
	{
		for (int iColm = 0; (iColm + columna) < anchura; iColm++)
		{
			if ((iColm + columna + 1) == anchura)
			{
				tablero[(iFila*anchura) + (iColm + columna)] = generarJewel(dificultad);
			}
			else {
				tablero[(iFila*anchura) + (iColm + columna)] = tablero[(iFila*altura) + (iColm + columna + 1)];
			}
		}
	}
}
void bombaRotarCPU(float* tablero, int anchura, int altura, int fila, int columna)
{
	float aux[9];
	int index = 0;
	for (int iColm = columna - 1; iColm <= columna + 1; iColm++)
	{
		for (int iFila = fila + 1; iFila >= fila - 1; iFila--)
		{
			aux[index] = tablero[iFila*anchura + iColm];
			index++;
		}
	}
	index = 0;
	for (int iFila = 0; iFila < 3; iFila++)
	{
		for (int iColumna = 0; iColumna < 3; iColumna++)
		{
			tablero[(iFila + fila - 1)*anchura + (columna - 1) + iColumna] = aux[index];
			index++;
		}
	}
}
int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int size;

	char ficheroGuardado[9] = "save.txt";

	float *tablero;
	float* tablero_d;
	bool jugando = true;

	int eleccion = 2;
	bool encontrado = false;
	std::cout << "Desea cargar una partida guardada? 1.-SI   2.-NO\n";
	std::cin >> eleccion;
	if (eleccion == 1)
	{
		encontrado = precargar(anchura, altura, dificultad, ficheroGuardado);
		std::cout << "Cargando Tablero de " << anchura << "x" << altura << " con dificultad: " << dificultad;
		std::cout << std::endl;
	}
	if (!encontrado || (eleccion == 2))
	{
		std::cout << "Anchura del tablero: ";
		std::cin >> anchura;

		std::cout << "Altura del tablero: ";
		std::cin >> altura;

		std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil\n";
		std::cin >> dificultad;
	}
	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO\n";
	std::cin >> seleccion;

	switch (seleccion) {
	case 1: automatico = true; break;
	case 2: automatico = false; break;
	default: printf("Valor no valido.\n"); return -1;
	}
	size = anchura*altura;
	tablero = (float*)malloc(size * sizeof(float));
	hipMalloc((void**)&tablero_d, size);
	//Se inicializa la matriz
	if (encontrado)
	{
		cargar(anchura, altura, tablero, ficheroGuardado);
		std::cout << "Se ha cargado el Tablero: \n";
	}
	else {
		generacionInicialRandomJewels(tablero, dificultad, anchura, altura);
		std::cout << "Se crea un tablero nuevo: \n";
	}

	//Tama�o de los bloques a crear en CUDA
	int TILE_WIDTH = obtenerTileWidth(anchura, altura);

	//Bucle principal del juego
	while (jugando) {

		//Se muestra el tablero
		printTablero(tablero, anchura, altura);

		int jewel1_x = 0;
		int jewel1_y = 0;
		int accion = 0;

		//Se deja escoger eleccion, si es automatico hay que intercambiar cualquier jewel primero, aunque no se haga
		std::cout << "Acci�n a realizar:\n";
		std::cout << "(1) Intercambiar Jewels\n";
		std::cout << "(2) Usar una Bomba\n";
		std::cout << "(3) Guardar partida\n";
		std::cout << "(4) Exit\n";
		std::cout << "Elija accion: ";

		std::cin >> accion;


		switch (accion) {
		case 1: {

			std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
			std::cout << "X: ";
			std::cin >> jewel1_x;
			std::cout << "Y: ";
			std::cin >> jewel1_y;

			if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
				printf("Posicion erronea.\n");
				continue;
			}

			int direccion = 0;
			std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
			std::cin >> direccion;

			if (direccion > 4 && direccion > 1) {
				printf("Direccion erronea.\n");
				continue;
			}
			else {
				switch (direccion)
				{
				case 1: //Arriba
				{
					if (jewel1_y == altura)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 2: //Abajo
				{
					if (jewel1_y == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 3: //Izquierda
				{
					if (jewel1_x == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 4: //Derecha
				{
					if (jewel1_x == anchura - 1)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				}

				if (seleccion == 1)
					analisisTableroAutomatico(dificultad, tablero, anchura, altura, TILE_WIDTH);
				else
					intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura, seleccion, dificultad, TILE_WIDTH);
			}

			break;
		}
		case 2: {
			// Bomba
			int bomba = 0;
			int fila = 0, columna = 0;
			std::cout << "Elija una bomba:";

			/* Bombas por tipo de dificultad */
			switch (dificultad) {
			case 1: {
				std::cout << "(1) Bomba de fila ";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				std::cout << "X: ";
				std::cin >> fila;
				bombaFila(tablero, anchura, altura, dificultad, fila);
				break;
			}
			case 2: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				}
				break;
			}
			case 3: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "(3) Bomba de rotacion 3x3 (la jewel elegida es el centro)";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				case 3:
				{
					std::cout << "X: ";
					std::cin >> fila;
					std::cout << "Y: ";
					std::cin >> columna;
					if ((fila - 1) < 0 || (fila + 1) >= altura || (columna - 1) < 0 || (columna + 1) >= anchura)
					{
						std::cout << "Rotacion no valida" << std::endl;
					}
					else
					{
						bombaRotarCPU(tablero, anchura, altura, fila, columna);
					}
					break;
				}
				}
				break;
			}
			}
			break;
		}
		case 3: {
			guardado(tablero, anchura, altura, dificultad, ficheroGuardado);
			std::cout << "Guardado correcto.\n";
			break;
		}
		case 4:
		{
			free(tablero);
			hipFree(tablero_d);
			return 0;
		}
		}

	}

	free(tablero);
	hipFree(tablero_d);
	return 0;
}