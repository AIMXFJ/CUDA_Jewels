#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <fstream>

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
	return -1;
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	srand(time(NULL));
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}
void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j + i*anchura]);
		}
	}
	printf("\n");
}

//TODO VERTICAL
/*__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d, int dificultad, int anchura, int altura, int final) {
	//int ty = threadIdx.x + jewels_eliminadas_d[1];
	//int tx = threadIdx.y + jewels_eliminadas_d[0];
	int tx = threadIdx.y;

	printf("\ntx:%i\n",tx);

	printf("\n eliminadas jewels _Device -> ");
	for (int q = 0; q < final; q++) {
		printf("%f |", jewels_eliminadas_d[q]);
	}
	printf("\n");

	//Horizontal
	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2]) {
		int posicion_abajo = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2) + 1] - 1) * anchura;
		int posicion = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2) + 1]) * anchura;

		printf("\ntx*2:%i  tx*2+1:%i\n", tx * 2, tx * 2 + 1);
		printf("\nposiciones x:%f y:%f\n", jewels_eliminadas_d[tx * 2], jewels_eliminadas_d[tx * 2 + 1]);

		if (jewels_eliminadas_d[(tx * 2) + 1]-1 < altura) {
			tablero_d[posicion_abajo] = tablero_d[posicion];
			tablero_d[posicion] = -1;
		}
		else {
			if (jewels_eliminadas_d[(tx * 2) + 1]-1 == altura) {
				tablero_d[posicion_abajo] = -1;
			}
		}
	}//Vertical
	else {
		int posicion_arriba = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2) + 1] - 1 + final/2) * anchura;
		int posicion = jewels_eliminadas_d[tx * 2] + (jewels_eliminadas_d[(tx * 2)+1] - 1) * anchura;

		if (jewels_eliminadas_d[(tx * 2) + 1] - 1 + final / 2 < altura) {
			tablero_d[posicion] = tablero_d[posicion_arriba];
			tablero_d[posicion_arriba] = -1;
		}
		else {
			if (jewels_eliminadas_d[(tx * 2) + 1] == altura - 1)
				jewels_eliminadas_d[(tx * 2) + 1] == -1;
		}

		//float value = tablero_d[tx + (ty)*anchura];
		//tablero_d[tx + (ty - final / 2)*(anchura)] = value;
		//tablero_d[tx + (ty)*anchura] = -1;
	}

	//tablero_d[tx+(ty-1)*anchura]=tablero_d[tx + ty*anchura];
	//tablero_d[tx + ty*anchura] = -1;
}*/

//TODO: Se pisan las filas entre ellas al no ir en orden.
/*__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d,int dificultad, int anchura, int altura, int final) {
	int tx = threadIdx.x + jewels_eliminadas_d[0];
	int ty = blockIdx.y + jewels_eliminadas_d[1];
	printf("\nBidx x:%i y:%i  | thrdIdx x:%i y:%i\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	printf("\nFinal: %i\n", final);

	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2]) {
		//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
			//printf("A");
			//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
				printf("\THREAD X:%i  Y:%i\n", tx, ty);
				if (ty + 1 < altura) {
					//if ty + 1 == altura
					float value = tablero_d[tx + (ty + 1)*anchura];

					__syncthreads();

					tablero_d[tx + (ty)*(anchura)] = value;

					__syncthreads();

					tablero_d[tx + (ty + 1)*anchura] = -1;
				}
				else {
						//tablero_d[tx + ty*anchura] = -2;
				}
		//	}
	//	}
	}
	else {
		//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
			//printf("A");
			//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
				//printf("\nBUCLE X:%i  Y:%i\n", x, y);
				if (ty < altura) {
					if (ty >= jewels_eliminadas_d[final - 2]) {
						float value = tablero_d[tx + (ty)*anchura];
						tablero_d[tx + (ty - final / 2)*(anchura)] = value;
						tablero_d[tx + (ty)*anchura] = -1;
					}
					else {
						tablero_d[tx + (ty)*anchura] = -1;
					}
				}
			//}
		//}
	}

	/*if (altura >= anchura) max = altura;
	else max = anchura;
	printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);

	for (int i = 0; i < max; i++) {
		if ((tx == jewels_eliminadas_d[i]) && (ty)>(jewels_eliminadas_d[i + 1])) {
			tablero_d[tx + (ty - 1)*(anchura)] = tablero_d[tx + ty*anchura];
		}

		if (ty == altura) {
			//Generar jewel random
			hiprandState state;

			hiprand_init((unsigned long long)clock(), i, 0, &state);

			tablero_d[tx + ty*anchura] = hiprand_uniform(&state);
		}
		i++;
	}*
}*/

//Elimina las jewels recibidas, bajas las filas para rellenas, y genera arriba del todo jewels nuevas. TODO
/*void eliminarJewels(float* tablero, float* jewels_eliminadas,int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *aux;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	aux = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < 2*max; i++) {
		aux[i]=jewels_eliminadas[i];
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
	//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
	int final = 0;

	for (int i = 0; i < max * 2; i++) {
		printf("\ni:%i valor:%f\n", i, jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max * 2;

	//Configuracion de ejecucion, 1 bloque por fila con tantos hilos como columnas
	//dim3 dimBlock(altura-jewels_eliminadas[1]-1,1);
	//dim3 dimGrid(1,jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);

	//n� de bloques
	dim3 dimGrid(1,1);

	printf("\nfinal: %i\n",final);
	printf("\n");
	for (int w = 0; w < final; w++) {
		printf("%f |",aux[w]);
	}
	printf("\n");

	if(aux[0]>=0)
	for (int z = 1; z <= altura-aux[1]-1; z++) {
		printf("\nantes buc k <= %f\n", aux[final - 2] - aux[0]);
		for (int k = 0; k < final; k+=2) {
			jewels_eliminadas[k] = aux[k];
			jewels_eliminadas[k + 1] = aux[k + 1] + z;
			printf("\na�adido a eliminadas x:%f y:%f\n",aux[k],aux[k+1]+z);
		}

		printf("\n eliminadas jewels -> ");
		for (int q = 0; q < final; q++) {
			printf("%f |", jewels_eliminadas[q]);
		}
		printf("\n");

		//Inicio del calculo, misma funcion de analisis en manual y automatico
		hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

		if (jewels_eliminadas[1] == jewels_eliminadas[3]) {
			dim3 dimBlock(1, jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);
			eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
		}
		else {
			dim3 dimBlock(altura - jewels_eliminadas[1] + 1, 1);
			eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
		}
		printf("\nLLAMADA\n");

		//Transfiere las jewels a eliminar de la GPU al host
		hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	}

	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}*/

__global__ void eliminarJewelsKernel(float* tablero_d, float* tablero_aux_d, float* jewels_eliminadas_d, int dificultad, int anchura, int altura, int final) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	//printf("\nBidx x:%i y:%i  | thrdIdx x:%i y:%i\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//printf("\nFinal: %i\n", final);

	if (jewels_eliminadas_d[0] != jewels_eliminadas_d[2] && tx >= jewels_eliminadas_d[0] && tx <= jewels_eliminadas_d[final - 2] && ty >= jewels_eliminadas_d[1]) {
		//printf("\THREAD X:%i  Y:%i\n", tx, ty);
		if (ty + 1 < altura) {
			float value = tablero_aux_d[tx + (ty + 1)*anchura];

			//printf("\nvalue: %f\n",value);

			tablero_d[tx + (ty)*(anchura)] = value;

			//tablero_d[tx + (ty + 1)*anchura] = -1;
		}
		else {
			//printf("\nFin\n");
			tablero_d[tx + ty*anchura] = -1;
		}
	}
	else {
		if (ty < altura && tx == jewels_eliminadas_d[0] && ty > jewels_eliminadas_d[1]) {
			float value = tablero_aux_d[tx + (ty)*anchura];
			tablero_d[tx + (ty - final / 2)*(anchura)] = value;
			//tablero_d[tx + (ty)*anchura] = -1;
		}
		if (ty >= altura - final / 2 && ty < altura && tx == jewels_eliminadas_d[0]) {
			tablero_d[tx + (ty)*anchura] = -1;
		}
	}
}

void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *tablero_aux_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&tablero_aux_d, size);
	hipMemcpy(tablero_aux_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	dim3 dimGrid(1, 1);
	dim3 dimBlock(anchura, altura);
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

	int final = 0;

	for (int i = 0; i < max * 2; i++) {
		printf("\ni:%i valor:%f\n", i, jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max * 2;

	eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, tablero_aux_d, jewels_eliminadas_d, dificultad, anchura, altura, final);

	hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	for (int k = 0; k < size; k++) {
		if (tablero[k] == -1) {
			srand(time(NULL));
			switch (dificultad) {
			case 1: {
				int randJewel = rand() % 4 + 1;
				tablero[k] = randJewel;
				break;
			}
			case 2: {
				int randJewel = rand() % 6 + 1;
				tablero[k] = randJewel;
				break;
			}
			case 3: {
				int randJewel = rand() % 8 + 1;
				tablero[k] = randJewel;
				break;
			}
			};
		}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
	hipFree(tablero_aux_d);
}

/*void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	float *aux;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	aux = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < 2 * max; i++) {
		aux[i] = jewels_eliminadas[i];
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 2 * max * sizeof(float));

	//for (int y = jewels_eliminadas_d[1]; y < altura; y++) {
	//for (int x = jewels_eliminadas_d[0]; x <= jewels_eliminadas_d[final - 2]; x++) {
	int final = 0;

	for (int i = 0; i < max * 2; i++) {
		printf("\ni:%i valor:%f\n", i, jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max * 2;

	//Configuracion de ejecucion, 1 bloque por fila con tantos hilos como columnas
	//dim3 dimBlock(altura-jewels_eliminadas[1]-1,1);
	//dim3 dimGrid(1,jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);

	//n� de bloques
	dim3 dimGrid(1, 1);

	printf("\nfinal: %i\n", final);
	printf("\n");
	for (int w = 0; w < final; w++) {
		printf("%f |", aux[w]);
	}
	printf("\n");

	if (aux[0] >= 0)
		for (int z = 1; z <= altura - aux[1] - 1; z++) {
			printf("\nantes buc k <= %f\n", aux[final - 2] - aux[0]);
			for (int k = 0; k < final; k += 2) {
				jewels_eliminadas[k] = aux[k];
				jewels_eliminadas[k + 1] = aux[k + 1] + z;
				printf("\na�adido a eliminadas x:%f y:%f\n", aux[k], aux[k + 1] + z);
			}

			printf("\n eliminadas jewels -> ");
			for (int q = 0; q < final; q++) {
				printf("%f |", jewels_eliminadas[q]);
			}
			printf("\n");

			//Inicio del calculo, misma funcion de analisis en manual y automatico
			hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 2 * max * sizeof(float), hipMemcpyHostToDevice);

			if (jewels_eliminadas[1] == jewels_eliminadas[3]) {
				dim3 dimBlock(1, jewels_eliminadas[final - 2] - jewels_eliminadas[0] + 1);
				eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
			}
			else {
				dim3 dimBlock(altura - jewels_eliminadas[1] + 1, 1);
				eliminarJewelsKernel << <dimGrid, dimBlock >> > (tablero_d, jewels_eliminadas_d, dificultad, anchura, altura, final);
			}
			printf("\nLLAMADA\n");

			//Transfiere las jewels a eliminar de la GPU al host
			hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

		}

	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		tablero_d[tx + (ty + 1)*anchura] = randJewel;
		break;
	}
	}

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}*/

__global__ void analisisTableroAutomaticoKernel(float *tablero_d, float *aux_d, int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int jewels_posibles_der = 0;

	//Si tiene por la derecha
	if ((tx + 2) < anchura) {
		if (((tx + 2) + ty*anchura <= altura*anchura) && tablero_d[tx + 2 + ty*anchura] == tablero_d[tx + ty*anchura]) {
			int i = 2;
			while ((tx + i + ty*anchura <= altura*anchura) && tablero_d[tx + i + ty*anchura] == tablero_d[tx + ty*anchura]) {
				jewels_posibles_der++;
				i++;
			}

			aux_d[tx + ty*anchura] = jewels_posibles_der + 1;
		}
		else {
			aux_d[tx + ty*anchura] = 1;
		}
	}
	else {
		aux_d[tx + ty*anchura] = 1;
	}
}

void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	int jewels_posibles_izq = 0;
	int jewels_posibles_der = 0;
	//printf("\nHORIZONTAL\n");
	//Si tiene por la izquierda
	if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x - i + y*anchura >= 0) && (x - i >= 0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_izq++;
			i++;
		}
	}

	//Si tiene por la derecha
	if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x + i + y*anchura <= size) && (x + i < anchura) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_der++;
			i++;
		}
	}

	//Se pueden eliminar horizontalmente
	if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
		int salto = 0;

		//printf("\nIZQ:%i   DER:%i\n",jewels_posibles_izq,jewels_posibles_der);

		for (int j = jewels_posibles_izq; j >= (1); j--) {
			jewels_eliminadas[salto] = x - j;
			jewels_eliminadas[salto + 1] = y;
			salto += 2;
		}

		jewels_eliminadas[jewels_posibles_izq * 2] = x;
		jewels_eliminadas[jewels_posibles_izq * 2 + 1] = y;

		salto = 2;
		for (int k = 1; k <= jewels_posibles_der; k++) {
			jewels_eliminadas[salto + jewels_posibles_izq * 2] = x + k;
			jewels_eliminadas[salto + jewels_posibles_izq * 2 + 1] = y;
			salto += 2;
		}
	}
	else {	//Analizamos la vertical
		int jewels_posibles_arrib = 0;
		int jewels_posibles_abaj = 0;

		//printf("\nVERTICAL\n");
		//Si tiene por abajo
		if ((x + (y - 1)*anchura >= 0) && tablero[x + (y - 1)*anchura] == tablero[x + y*anchura]) {
			printf("\nABAJO\n");
			int i = 1;
			while ((x + (y - i)*anchura >= 0) && tablero[x + (y - i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_abaj++;
				//printf("\nTIENE ABAJO\n");
				i++;
			}
		}

		//Si tiene por arriba
		if ((x + 1 + y*anchura <= size) && tablero[x + (y + 1)*anchura] == tablero[x + y*anchura]) {
			//printf("\nARRIBA\n");
			int i = 1;
			while ((x + (y + i)*anchura <= size) && tablero[x + (y + i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_arrib++;
				//printf("\nTIENE ARRIBA\n");
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {
			//printf("\nSE PUEDE\n");

			int salto = 0;
			for (int j = jewels_posibles_abaj; j >= (1); j--) {
				jewels_eliminadas[salto] = x;
				jewels_eliminadas[salto + 1] = y - j;
				salto += 2;
			}

			jewels_eliminadas[jewels_posibles_abaj * 2] = x;
			jewels_eliminadas[jewels_posibles_abaj * 2 + 1] = y;

			salto = 2;
			for (int k = 1; k <= jewels_posibles_arrib; k++) {
				jewels_eliminadas[salto + jewels_posibles_abaj * 2] = x;
				jewels_eliminadas[salto + jewels_posibles_abaj * 2 + 1] = y + k;
				salto += 2;
			}
		}
	}

	for (int q = 0; q < 2 * max; q++) {
		if (q % 2 != 0) {
			printf(" y:%f\n", jewels_eliminadas[q]);
		}
		else {
			printf("| x:%f\n", jewels_eliminadas[q]);
		}
	}
	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura, int seleccion, int dificultad) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion)
	{
	case 1: //Arriba
	{
		jewel2_y += 1;
		break;
	}
	case 2: //Abajo
	{
		jewel2_y -= 1;
		break;
	}
	case 3: //Izquierda
	{
		jewel2_x -= 1;
		break;
	}
	case 4: //Derecha
	{
		jewel2_x += 1;
		break;
	}
	}
	int aux1;

	aux1 = tablero[jewel2_x + jewel2_y*anchura];

	tablero[jewel2_x + jewel2_y*anchura] = tablero[jewel1_x + jewel1_y*anchura];
	tablero[jewel1_x + jewel1_y*anchura] = aux1;

	analisisTableroManual(dificultad, tablero, anchura, altura, jewel2_x, jewel2_y);
}

//CUDA CPU Function. Analiza la mejor opcion y la ejecuta
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *aux_d;
	float *aux;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));
	aux = (float*)malloc(size);

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	for (int p = 0; p < size; p++) {
		aux[p] = 1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&aux_d, size);
	hipMemcpy(aux_d, aux, size, hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	analisisTableroAutomaticoKernel <<<dimGrid,dimBlock>>> (tablero_d, aux_d, dificultad, anchura, altura);
	if (hipSuccess != hipGetLastError())
		printf("\nCUDA Error!\n");

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(aux, aux_d, size, hipMemcpyDeviceToHost);

	printTablero(aux, anchura, altura);

	int x_mejor = 0;
	int y_mejor = 0;
	int valor_mejor = 0;

	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			if (aux[x+y*anchura] > valor_mejor) {
				valor_mejor = aux[x+y*anchura];
				x_mejor = x;
				y_mejor = y;
			}
		}
	}

	if (valor_mejor >= 3) {
		intercambiarPosiciones(tablero, x_mejor, y_mejor, 4, anchura, altura, 1, dificultad);
	}
}

bool precargar(int& anchura, int& altura, int& dificultad, char* fichero)
{
	std::ifstream fCarga(fichero);
	char tam[4];
	if (!fCarga.is_open())
	{
		std::cout << "ERROR: no existe un archivo guardado." << std::endl;
		return false;
	}

	fCarga.getline(tam, 4);

	anchura = (int)tam[0] - 48;
	altura = (int)tam[1] - 48;
	dificultad = (int)tam[2] - 48;

	fCarga.close();
	return true;
}
void cargar(int anchura, int altura, float*  tablero, char* fichero)
{
	char* array = (char*)malloc(anchura*altura + 1 + 3);
	std::ifstream fCarga(fichero);
	fCarga.getline(array, (anchura*altura + 1 + 3));
	for (int i = 0; i < anchura*altura; i++)
	{
		tablero[i] = array[i + 3] - 48;
	}
	free(array);
	fCarga.close();
}

void guardado(float* tablero, int anchura, int altura, int dificultad, char* fichero)
{
	//Sistema de guardado
	std::ofstream ficheroGuardado;
	ficheroGuardado.open(fichero);
	ficheroGuardado.clear();
	/* Almacenar anchura y altura*/
	ficheroGuardado << anchura;
	ficheroGuardado << altura;
	ficheroGuardado << dificultad;
	/* Almacenar Resto */
	for (int index = 0; index < anchura*altura; index++)
	{
		ficheroGuardado << tablero[index];
	}
	ficheroGuardado.close();
}
void bombaFila(float* tablero, int anchura, int altura, int dificultad, int fila) {

	for (int iFila = 0; (iFila + fila) < altura; iFila++)
	{
		for (int iColm = 0; iColm < anchura; iColm++)
		{
			if ((iFila + fila + 1) < altura)
			{
				tablero[(iFila + fila)*anchura + iColm] = tablero[(iFila + fila + 1)*altura + iColm];
			}
			else {
				tablero[(iFila + fila)*anchura + iColm] = generarJewel(dificultad);
			}
		}
	}
}

void bombaColumna(float* tablero, int anchura, int altura, int dificultad, int columna) {

	for (int iFila = 0; iFila < altura; iFila++)
	{
		for (int iColm = 0; (iColm + columna) < anchura; iColm++)
		{
			if ((iColm + columna + 1) == anchura)
			{
				tablero[(iFila*anchura) + (iColm + columna)] = generarJewel(dificultad);
			}
			else {
				tablero[(iFila*anchura) + (iColm + columna)] = tablero[(iFila*altura) + (iColm + columna + 1)];
			}
		}
	}
}
void bombaRotarCPU(float* tablero, int anchura, int altura, int fila, int columna)
{
	float aux[9];
	int index = 0;
	for (int iColm = columna - 1; iColm <= columna + 1; iColm++)
	{
		for (int iFila = fila + 1; iFila >= fila - 1; iFila--)
		{
			aux[index] = tablero[iFila*anchura + iColm];
			index++;
		}
	}
	index = 0;
	for (int iFila = 0; iFila < 3; iFila++)
	{
		for (int iColumna = 0; iColumna < 3; iColumna++)
		{
			tablero[(iFila + fila - 1)*anchura + (columna - 1) + iColumna] = aux[index];
			index++;
		}
	}
}
int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int TILE_WIDTH = 16;
	int size;

	char ficheroGuardado[9] = "save.txt";

	float *tablero;
	float* tablero_d;
	bool jugando = true;

	int eleccion = 2;
	bool encontrado = false;
	std::cout << "Desea cargar una partida guardada? 1.-SI   2.-NO\n";
	std::cin >> eleccion;
	if (eleccion == 1)
	{
		encontrado = precargar(anchura, altura, dificultad, ficheroGuardado);
		std::cout << "Cargando Tablero de " << anchura << "x" << altura << " con dificultad: " << dificultad;
		std::cout << std::endl;
	}
	if (!encontrado || (eleccion == 2))
	{
		std::cout << "Anchura del tablero: ";
		std::cin >> anchura;

		std::cout << "Altura del tablero: ";
		std::cin >> altura;

		std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil\n";
		std::cin >> dificultad;
	}
	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO\n";
	std::cin >> seleccion;

	switch (seleccion) {
	case 1: automatico = true; break;
	case 2: automatico = false; break;
	default: printf("Valor no valido.\n"); return -1;
	}
	size = anchura*altura;
	tablero = (float*)malloc(size * sizeof(float));
	hipMalloc((void**)&tablero_d, size);
	//Se inicializa la matriz
	if (encontrado)
	{
		cargar(anchura, altura, tablero, ficheroGuardado);
		std::cout << "Se ha cargado el Tablero: \n";
	}
	else {
		generacionInicialRandomJewels(tablero, dificultad, anchura, altura);
		std::cout << "Se crea un tablero nuevo: \n";
	}
	//Bucle principal del juego
	while (jugando) {

		printTablero(tablero, anchura, altura);

		int jewel1_x = 0;
		int jewel1_y = 0;
		int accion = 0;

		std::cout << "Acci�n a realizar:\n";
		std::cout << "(1) Intercambiar Jewels\n";
		std::cout << "(2) Usar una Bomba\n";
		std::cout << "(3) Guardar partida\n";
		std::cout << "(4) Exit\n";
		std::cout << "Elija accion: ";

		std::cin >> accion;


		switch (accion) {
		case 1: {

			std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
			std::cout << "X: ";
			std::cin >> jewel1_x;
			std::cout << "Y: ";
			std::cin >> jewel1_y;

			if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
				printf("Posicion erronea.\n");
				continue;
			}

			int direccion = 0;
			std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
			std::cin >> direccion;

			if (direccion > 4 && direccion > 1) {
				printf("Direccion erronea.\n");
				continue;
			}
			else {
				switch (direccion)
				{
				case 1: //Arriba
				{
					if (jewel1_y == altura)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 2: //Abajo
				{
					if (jewel1_y == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 3: //Izquierda
				{
					if (jewel1_x == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 4: //Derecha
				{
					if (jewel1_x == anchura - 1)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				}

				if (seleccion == 1)
					analisisTableroAutomatico(dificultad, tablero, anchura, altura);
				else
					intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura, seleccion, dificultad);
			}

			break;
		}
		case 2: {
			// Bomba
			int bomba = 0;
			int fila = 0, columna = 0;
			std::cout << "Elija una bomba:";

			/* Bombas por tipo de dificultad */
			switch (dificultad) {
			case 1: {
				std::cout << "(1) Bomba de fila ";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				std::cout << "X: ";
				std::cin >> fila;
				bombaFila(tablero, anchura, altura, dificultad, fila);
				break;
			}
			case 2: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				}
				break;
			}
			case 3: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "(3) Bomba de rotacion 3x3 (la jewel elegida es el centro)";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				case 3:
				{
					std::cout << "X: ";
					std::cin >> fila;
					std::cout << "Y: ";
					std::cin >> columna;
					if ((fila - 1) < 0 || (fila + 1) >= altura || (columna - 1) < 0 || (columna + 1) >= anchura)
					{
						std::cout << "Rotacion no valida" << std::endl;
					}
					else
					{
						bombaRotarCPU(tablero, anchura, altura, fila, columna);
					}
					break;
				}
				}
				break;
			}
			}
			break;
		}
		case 3: {
			guardado(tablero, anchura, altura, dificultad, ficheroGuardado);
			std::cout << "Guardado correcto.\n";
			break;
		}
		case 4:
		{
			free(tablero);
			hipFree(tablero_d);
			return 0;
		}
		}

	}

	free(tablero);
	hipFree(tablero_d);
	return 0;
}