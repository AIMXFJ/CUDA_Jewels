#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <fstream>

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
	return -1;
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}

void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j + i*anchura]);
		}
	}
	printf("\n");
}

//TODO: revisar
void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	int final = 0;
	
	for (int i = 0; i < max; i++) {
		printf("\ni:%i valor:%i\n",i,jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
		}
	}

	printf("\nFinal: %i\n", final);

	for (int y = jewels_eliminadas[1]; y < altura; y++) {
		for (int x = jewels_eliminadas[0]; x < jewels_eliminadas[final-1]; x++) {
			printf("\nBUCLE X:%I  Y:%I\n",x,y);
				tablero[x + (y - 1)*(anchura)] = tablero[x + y*anchura];

				if (y == altura) {
					//Generar jewel random
					printf("\nGenerando\n");
					tablero[x + y*anchura] = -1;
					/*switch (dificultad) {
					case 1: {
						int randJewel = rand() % 4 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					case 2: {
						int randJewel = rand() % 6 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					case 3: {
						int randJewel = rand() % 8 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					}*/
				}
		}
	}
}

//Funcion CPU. FUNCIONA
void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tiene posibles eliminables por la derecha o izquierda
	// (((x-1+y*anchura>=0)&&tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) || ((x+1+y*anchura<=size)&&tablero[x + 1 + y*anchura] == tablero[x + y*anchura])) {
	int jewels_posibles_izq = 0;
	int jewels_posibles_der = 0;
	printf("\nHORIZONTAL\n");
	//Si tiene por la izquierda
	if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x - i + y*anchura >= 0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_izq++;
			i++;
		}
	}

	//Si tiene por la derecha
	if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x + i + y*anchura <= size) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_der++;
			i++;
		}
	}

	//Se pueden eliminar horizontalmente
	if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
		int salto = 0;

		for (int j = jewels_posibles_izq; j >= (1); j--) {
			jewels_eliminadas[salto] = x - j;
			jewels_eliminadas[salto + 1] = y;
			salto += 2;
		}

		jewels_eliminadas[jewels_posibles_izq*2] = x;
		jewels_eliminadas[jewels_posibles_izq*2+1] = y;

		salto = 2;
		for (int k = 1; k <= jewels_posibles_der; k++) {
			jewels_eliminadas[salto + 1 + jewels_posibles_izq*2] = x + k;
			jewels_eliminadas[salto + 1 + jewels_posibles_izq*2 + 1] = y;
			salto += 2;
		}
	}
	else {	//Analizamos la vertical
		int jewels_posibles_arrib = 0;
		int jewels_posibles_abaj = 0;

		printf("\nVERTICAL\n");
		//Si tiene por abajo
		if ((x + (y - 1)*anchura >= 0) && tablero[x + (y - 1)*anchura] == tablero[x + y*anchura]) {
			printf("\nABAJO\n");
			int i = 1;
			while ((x + (y - i)*anchura >= 0) && tablero[x + (y - i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_abaj++;
				printf("\nTIENE ABAJO\n");
				i++;
			}
		}

		//Si tiene por arriba
		if ((x + 1 + y*anchura <= size) && tablero[x + (y + 1)*anchura] == tablero[x + y*anchura]) {
			printf("\nARRIBA\n");
			int i = 1;
			while ((x + (y + i)*anchura <= size) && tablero[x + (y + i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_arrib++;
				printf("\nTIENE ARRIBA\n");
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {
			printf("\nSE PUEDE\n");

			int salto = 0;
			for (int j = jewels_posibles_abaj; j >= (1); j++) {
				jewels_eliminadas[salto] = x;
				jewels_eliminadas[salto + 1] = y - j;
				salto += 2;
			}

			jewels_eliminadas[jewels_posibles_abaj*2] = x;
			jewels_eliminadas[jewels_posibles_abaj*2+1] = y;

			salto = 2;
			for (int k = 1; k <= jewels_posibles_arrib; k++) {
				jewels_eliminadas[salto + jewels_posibles_abaj*2] = x;
				jewels_eliminadas[salto + 1 + jewels_posibles_abaj*2 + 1] = y + k;
				salto += 2;
			}
		}
	}

	//("\nJewels a eliminar horizontal: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_3] / anchura, jewels_eliminadas[4], jewels_eliminadas[5] / anchura);
	for (int q = 0; q < 2 * max; q++) {
		if (q % 2 != 0) {
			printf(" y:%f\n", jewels_eliminadas[q]);
		}
		else {
			printf("| x:%f\n", jewels_eliminadas[q]);
		}
	}
	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura, int seleccion, int dificultad) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion)
	{
	case 1: //Arriba
	{
		jewel2_y += 1;
		break;
	}
	case 2: //Abajo
	{
		jewel2_y -= 1;
		break;
	}
	case 3: //Izquierda
	{
		jewel2_x -= 1;
		break;
	}
	case 4: //Derecha
	{
		jewel2_x += 1;
		break;
	}
	}
	int aux1;

	aux1 = tablero[jewel2_x + jewel2_y*anchura];

	tablero[jewel2_x + jewel2_y*anchura] = tablero[jewel1_x + jewel1_y*anchura];
	tablero[jewel1_x + jewel1_y*anchura] = aux1;

	if (seleccion == 2)
		analisisTableroManual(dificultad, tablero, anchura, altura, jewel2_x, jewel2_y);
}

//Funcion CPU. TODO: Arreglar calculo de contiguos, posible fallo al contar
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura) {
	int max = 0;
	int size = anchura*altura;
	int jewels_posibles_der = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	//Tablero auxiliar para la toma del mejor caso
	float* aux_tablero = (float*)malloc(altura * anchura * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	printf("\nAUTOMATICO\n");

	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			jewels_posibles_der = 0;

			//Si tiene por la derecha
			if ((x + 2) < anchura) {
				if (((x + 2) + y*anchura <= size) && tablero[x + 2 + y*anchura] == tablero[x + y*anchura]) {
					int i = 2;
					while ((x + i + y*anchura <= size) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
						jewels_posibles_der++;
						i++;
					}

					aux_tablero[x + y*anchura] = jewels_posibles_der + 1;
				}
				else {
					aux_tablero[x + y*anchura] = 1;
				}
			}
			else {
				aux_tablero[x + y*anchura] = 1;
			}
		}
	}

	int x_mejor = 0;
	int y_mejor = 0;
	int valor_mejor = 0;

	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			if (aux_tablero[x + y*anchura] > valor_mejor) {
				x_mejor = x;
				y_mejor = y;
				valor_mejor = aux_tablero[x + y*anchura];
			}
		}
	}

	//printf("\nTablero Aux Automatico:\n");
	//printTablero(aux_tablero, anchura, altura);


	//printf("\nMejores valores: x:%i  y:%i  valor:%i\n",x_mejor,y_mejor,valor_mejor);

	intercambiarPosiciones(tablero, x_mejor + 1, y_mejor, 4, anchura, altura, 1, dificultad);

	//Se puede eliminar
	if (valor_mejor >= 3) {
		jewels_eliminadas[0] = x_mejor;
		jewels_eliminadas[1] = y_mejor;

		int salto = 2;

		for (int j = 1; j <= (valor_mejor); j++) {
			jewels_eliminadas[salto] = x_mejor + j;
			jewels_eliminadas[salto + 1] = y_mejor;
			salto += 2;
		}
	}

	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);


}

bool precargar(int& anchura, int& altura, int& dificultad, char* fichero)
{
	std::ifstream fCarga(fichero);
	char tam[4];
	if (!fCarga.is_open())
	{
		std::cout << "ERROR: no existe un archivo guardado." << std::endl;
		return false;
	}

	fCarga.getline(tam, 4);

	anchura = (int)tam[0] - 48;
	altura = (int)tam[1] - 48;
	dificultad = (int)tam[2] - 48;

	fCarga.close();
	return true;
}
void cargar(int anchura, int altura, float*  tablero, char* fichero)
{
	char* array = (char*)malloc(anchura*altura + 1 + 3);
	std::ifstream fCarga(fichero);
	fCarga.getline(array, (anchura*altura + 1 + 3));
	for (int i = 0; i < anchura*altura; i++)
	{
		tablero[i] = array[i + 3] - 48;
	}
	free(array);
	fCarga.close();
}

void guardado(float* tablero, int anchura, int altura, int dificultad, char* fichero)
{
	//Sistema de guardado
	std::ofstream ficheroGuardado;
	ficheroGuardado.open(fichero);
	ficheroGuardado.clear();
	/* Almacenar anchura y altura*/
	ficheroGuardado << anchura;
	ficheroGuardado << altura;
	ficheroGuardado << dificultad;
	/* Almacenar Resto */
	for (int index = 0; index < anchura*altura; index++)
	{
		ficheroGuardado << tablero[index];
	}
	ficheroGuardado.close();
}
void bombaFila(float* tablero, int anchura, int altura, int dificultad, int fila) {

	for (int iFila = 0; (iFila + fila) < altura; iFila++)
	{
		for (int iColm = 0; iColm < anchura; iColm++)
		{
			if ((iFila + fila + 1) < altura)
			{
				tablero[(iFila + fila)*anchura + iColm] = tablero[(iFila + fila + 1)*altura + iColm];
			}
			else {
				tablero[(iFila + fila)*anchura + iColm] = generarJewel(dificultad);
			}
		}
	}
}

void bombaColumna(float* tablero, int anchura, int altura, int dificultad, int columna) {

	for (int iFila = 0; iFila < altura; iFila++)
	{
		for (int iColm = 0; (iColm + columna) < anchura; iColm++)
		{
			if ((iColm + columna + 1) == anchura)
			{
				tablero[(iFila*anchura) + (iColm + columna)] = generarJewel(dificultad);
			}
			else {
				tablero[(iFila*anchura) + (iColm + columna)] = tablero[(iFila*altura) + (iColm + columna + 1)];
			}
		}
	}
}
void bombaRotarCPU(float* tablero, int anchura, int altura, int fila, int columna)
{
	float aux[9];
	int index = 0;
	for (int iColm = columna - 1; iColm <= columna + 1; iColm++)
	{
		for (int iFila = fila + 1; iFila >= fila - 1; iFila--)
		{
			aux[index] = tablero[iFila*anchura + iColm];
			index++;
		}
	}
	index = 0;
	for (int iFila = 0; iFila < 3; iFila++)
	{
		for (int iColumna = 0; iColumna < 3; iColumna++)
		{
			tablero[(iFila + fila - 1)*anchura + (columna - 1) + iColumna] = aux[index];
			index++;
		}
	}
}
int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int TILE_WIDTH = 16;
	int size;

	char ficheroGuardado[9] = "save.txt";

	float *tablero;
	float* tablero_d;
	bool jugando = true;

	int eleccion = 2;
	bool encontrado = false;
	std::cout << "Desea cargar una partida guardada? 1.-SI   2.-NO\n";
	std::cin >> eleccion;
	if (eleccion == 1)
	{
		encontrado = precargar(anchura, altura, dificultad, ficheroGuardado);
		std::cout << "Cargando Tablero de " << anchura << "x" << altura << " con dificultad: " << dificultad;
		std::cout << std::endl;
	}
	if (!encontrado || (eleccion == 2))
	{
		std::cout << "Anchura del tablero: ";
		std::cin >> anchura;

		std::cout << "Altura del tablero: ";
		std::cin >> altura;

		std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil\n";
		std::cin >> dificultad;
	}
	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO\n";
	std::cin >> seleccion;

	switch (seleccion) {
	case 1: automatico = true; break;
	case 2: automatico = false; break;
	default: printf("Valor no valido.\n"); return -1;
	}
	size = anchura*altura;
	tablero = (float*)malloc(size * sizeof(float));
	hipMalloc((void**)&tablero_d, size);
	//Se inicializa la matriz
	if (encontrado)
	{
		cargar(anchura, altura, tablero, ficheroGuardado);
		std::cout << "Se ha cargado el Tablero: \n";
	}
	else {
		generacionInicialRandomJewels(tablero, dificultad, anchura, altura);
		std::cout << "Se crea un tablero nuevo: \n";
	}
	//Bucle principal del juego
	while (jugando) {

		printTablero(tablero, anchura, altura);

		int jewel1_x = 0;
		int jewel1_y = 0;
		int accion = 0;

		std::cout << "Acci�n a realizar:\n";
		std::cout << "(1) Intercambiar Jewels\n";
		std::cout << "(2) Usar una Bomba\n";
		std::cout << "(3) Guardar partida\n";
		std::cout << "(4) Exit\n";
		std::cout << "Elija accion: ";

		std::cin >> accion;


		switch (accion) {
		case 1: {

			std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
			std::cout << "X: ";
			std::cin >> jewel1_x;
			std::cout << "Y: ";
			std::cin >> jewel1_y;

			if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
				printf("Posicion erronea.\n");
				continue;
			}

			int direccion = 0;
			std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
			std::cin >> direccion;

			if (direccion > 4 && direccion > 1) {
				printf("Direccion erronea.\n");
				continue;
			}
			else {
				switch (direccion)
				{
				case 1: //Arriba
				{
					if (jewel1_y == altura)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 2: //Abajo
				{
					if (jewel1_y == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 3: //Izquierda
				{
					if (jewel1_x == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 4: //Derecha
				{
					if (jewel1_x == anchura - 1)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				}

				intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura, seleccion, dificultad);

				if (seleccion == 1)
					analisisTableroAutomatico(dificultad, tablero, anchura, altura);
			}

			break;
		}
		case 2: {
			// Bomba
			int bomba = 0;
			int fila = 0, columna = 0;
			std::cout << "Elija una bomba:";

			/* Bombas por tipo de dificultad */
			switch (dificultad) {
			case 1: {
				std::cout << "(1) Bomba de fila ";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				std::cout << "X: ";
				std::cin >> fila;
				bombaFila(tablero, anchura, altura, dificultad, fila);
				break;
			}
			case 2: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				}
				break;
			}
			case 3: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "(3) Bomba de rotacion 3x3 (la jewel elegida es el centro)";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				case 3:
				{
					std::cout << "X: ";
					std::cin >> fila;
					std::cout << "Y: ";
					std::cin >> columna;
					if ((fila - 1) < 0 || (fila + 1) >= altura || (columna - 1) < 0 || (columna + 1) >= anchura)
					{
						std::cout << "Rotacion no valida" << std::endl;
					}
					else
					{
						bombaRotarCPU(tablero, anchura, altura, fila, columna);
					}
					break;
				}
				}
				break;
			}
			}
			break;
		}
		case 3: {
			guardado(tablero, anchura, altura, dificultad, ficheroGuardado);
			std::cout << "Guardado correcto.\n";
			break;
		}
		case 4:
		{
			free(tablero);
			hipFree(tablero_d);
			return 0;
		}
		}

	}

	free(tablero);
	hipFree(tablero_d);
	return 0;
}