#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <fstream>

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	srand(time(NULL));
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
	return -1;
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	srand(time(NULL));
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
		}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}

void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j + i*anchura]);
		}
	}
	printf("\n");
}

void eliminarJewels(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	int final = 0;
	
	for (int i = 0; i < max*2; i++) {
		//printf("\ni:%i valor:%f\n",i,jewels_eliminadas[i]);
		if (jewels_eliminadas[i] < 0) {
			final = i;
			break;
		}
	}

	if (final == 0) final = max*2;

	//printf("\nFinal: %i\n", final);
	srand(time(NULL));

	if (jewels_eliminadas[0] != jewels_eliminadas[2]) {
		for (int y = jewels_eliminadas[1]; y < altura; y++) {
			//printf("A");
			for (int x = jewels_eliminadas[0]; x <= jewels_eliminadas[final - 2]; x++) {
				//printf("\nBUCLE X:%i  Y:%i\n", x, y);
				if (y + 1 < altura) {
					tablero[x + (y)*(anchura)] = tablero[x + (y + 1)*anchura];
					switch (dificultad) {
					case 1: {
						int randJewel = rand() % 4 + 1;
						tablero[x + (y+1)*anchura] = randJewel;
						break;
					}
					case 2: {
						int randJewel = rand() % 6 + 1;
						tablero[x + (y+1)*anchura] = randJewel;
						break;
					}
					case 3: {
						int randJewel = rand() % 8 + 1;
						tablero[x + (y+1)*anchura] = randJewel;
						break;
					}
					}
				}
				else {
					switch (dificultad) {
					case 1: {
						int randJewel = rand() % 4 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					case 2: {
						int randJewel = rand() % 6 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					case 3: {
						int randJewel = rand() % 8 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					}
				}
			}
		}
	}else{
		int posicion = jewels_eliminadas[0] + jewels_eliminadas[1] * anchura;
		float valor = tablero[posicion];
		for (int y = jewels_eliminadas[1]; y < altura; y++) {
			//printf("A");
			for (int x = jewels_eliminadas[0]; x <= jewels_eliminadas[final - 2]; x++) {
				//printf("\nBUCLE X:%i  Y:%i\n", x, y);
				if (y < altura) {
					if (y >= jewels_eliminadas[final-2]) {
						tablero[x + (y-final/2)*(anchura)] = tablero[x + (y)*anchura];
						switch (dificultad) {
						case 1: {
							int randJewel = rand() % 4 + 1;
							tablero[x + (y)*anchura] = randJewel;
							break;
						}
						case 2: {
							int randJewel = rand() % 6 + 1;
							tablero[x + (y)*anchura] = randJewel;
							break;
						}
						case 3: {
							int randJewel = rand() % 8 + 1;
							tablero[x + (y)*anchura] = randJewel;
							break;
						}
						}
					}
					else {
						switch (dificultad) {
						case 1: {
							int randJewel = rand() % 4 + 1;
							tablero[x + (y)*anchura] = randJewel;
							break;
						}
						case 2: {
							int randJewel = rand() % 6 + 1;
							tablero[x + (y)*anchura] = randJewel;
							break;
						}
						case 3: {
							int randJewel = rand() % 8 + 1;
							tablero[x + (y)*anchura] = randJewel;
							break;
						}
						}
					}
				}
			}
		}
	}
}

void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	int jewels_posibles_izq = 0;
	int jewels_posibles_der = 0;
	//printf("\nHORIZONTAL\n");
	//Si tiene por la izquierda
	if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x - i + y*anchura >= 0) && (x -i>=0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_izq++;
			i++;
		}
	}

	//Si tiene por la derecha
	if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
		int i = 1;
		while ((x + i + y*anchura <= size) && (x + i < anchura) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
			jewels_posibles_der++;
			i++;
		}
	}

	//Se pueden eliminar horizontalmente
	if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
		int salto = 0;

		//printf("\nIZQ:%i   DER:%i\n",jewels_posibles_izq,jewels_posibles_der);

		for (int j = jewels_posibles_izq; j >= (1); j--) {
			jewels_eliminadas[salto] = x - j;
			jewels_eliminadas[salto + 1] = y;
			salto += 2;
		}

		jewels_eliminadas[jewels_posibles_izq*2] = x;
		jewels_eliminadas[jewels_posibles_izq*2+1] = y;

		salto = 2;
		for (int k = 1; k <= jewels_posibles_der; k++) {
			jewels_eliminadas[salto + jewels_posibles_izq*2] = x + k;
			jewels_eliminadas[salto + jewels_posibles_izq*2 + 1] = y;
			salto += 2;
		}
	}
	else {	//Analizamos la vertical
		int jewels_posibles_arrib = 0;
		int jewels_posibles_abaj = 0;

		//printf("\nVERTICAL\n");
		//Si tiene por abajo
		if ((x + (y - 1)*anchura >= 0) && tablero[x + (y - 1)*anchura] == tablero[x + y*anchura]) {
			printf("\nABAJO\n");
			int i = 1;
			while ((x + (y - i)*anchura >= 0) && tablero[x + (y - i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_abaj++;
				//printf("\nTIENE ABAJO\n");
				i++;
			}
		}

		//Si tiene por arriba
		if ((x + 1 + y*anchura <= size) && tablero[x + (y + 1)*anchura] == tablero[x + y*anchura]) {
			//printf("\nARRIBA\n");
			int i = 1;
			while ((x + (y + i)*anchura <= size) && tablero[x + (y + i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_arrib++;
				//printf("\nTIENE ARRIBA\n");
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {
			//printf("\nSE PUEDE\n");

			int salto = 0;
			for (int j = jewels_posibles_abaj; j >= (1); j--) {
				jewels_eliminadas[salto] = x;
				jewels_eliminadas[salto + 1] = y - j;
				salto += 2;
			}

			jewels_eliminadas[jewels_posibles_abaj*2] = x;
			jewels_eliminadas[jewels_posibles_abaj*2+1] = y;

			salto = 2;
			for (int k = 1; k <= jewels_posibles_arrib; k++) {
				jewels_eliminadas[salto + jewels_posibles_abaj*2] = x;
				jewels_eliminadas[salto + jewels_posibles_abaj*2 + 1] = y + k;
				salto += 2;
			}
		}
	}

	/*for (int q = 0; q < 2 * max; q++) {
		if (q % 2 != 0) {
			printf(" y:%f\n", jewels_eliminadas[q]);
		}
		else {
			printf("| x:%f\n", jewels_eliminadas[q]);
		}
	}*/
	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura, int seleccion, int dificultad) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion)
	{
	case 1: //Arriba
	{
		jewel2_y += 1;
		break;
	}
	case 2: //Abajo
	{
		jewel2_y -= 1;
		break;
	}
	case 3: //Izquierda
	{
		jewel2_x -= 1;
		break;
	}
	case 4: //Derecha
	{
		jewel2_x += 1;
		break;
	}
	}
	int aux1;

	aux1 = tablero[jewel2_x + jewel2_y*anchura];

	tablero[jewel2_x + jewel2_y*anchura] = tablero[jewel1_x + jewel1_y*anchura];
	tablero[jewel1_x + jewel1_y*anchura] = aux1;

	if (seleccion == 2)
		analisisTableroManual(dificultad, tablero, anchura, altura, jewel2_x, jewel2_y);
}

//Funcion CPU. TODO: Arreglar calculo de contiguos, posible fallo al contar
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura) {
	int max = 0;
	int size = anchura*altura;
	int jewels_posibles_der = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	//Tablero auxiliar para la toma del mejor caso
	float* aux_tablero = (float*)malloc(altura * anchura * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//printf("\nAUTOMATICO\n");

	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			jewels_posibles_der = 0;

			//Si tiene por la derecha
			if ((x + 2) < anchura) {
				if (((x + 2) + y*anchura <= size) && tablero[x + 2 + y*anchura] == tablero[x + y*anchura]) {
					int i = 2;
					while ((x + i + y*anchura <= size) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
						jewels_posibles_der++;
						i++;
					}

					aux_tablero[x + y*anchura] = jewels_posibles_der + 1;
				}
				else {
					aux_tablero[x + y*anchura] = 1;
				}
			}
			else {
				aux_tablero[x + y*anchura] = 1;
			}
		}
	}

	int x_mejor = 0;
	int y_mejor = 0;
	int valor_mejor = 0;

	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			if (aux_tablero[x + y*anchura] > valor_mejor) {
				x_mejor = x;
				y_mejor = y;
				valor_mejor = aux_tablero[x + y*anchura];
			}
		}
	}

	//printf("\nTablero Aux Automatico:\n");
	//printTablero(aux_tablero, anchura, altura);


	//printf("\nMejores valores: x:%i  y:%i  valor:%i\n",x_mejor,y_mejor,valor_mejor);

	intercambiarPosiciones(tablero, x_mejor, y_mejor, 4, anchura, altura, 1, dificultad);

	//Se puede eliminar
	if (valor_mejor >= 3) {
		jewels_eliminadas[0] = x_mejor;
		jewels_eliminadas[1] = y_mejor;

		int salto = 2;

		for (int j = 1; j <= (valor_mejor); j++) {
			jewels_eliminadas[salto] = x_mejor + j;
			jewels_eliminadas[salto + 1] = y_mejor;
			salto += 2;
		}
	}

	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);


}

bool precargar(int& anchura, int& altura, int& dificultad, char* fichero)
{
	std::ifstream fCarga(fichero);
	char tam[4];
	if (!fCarga.is_open())
	{
		std::cout << "ERROR: no existe un archivo guardado." << std::endl;
		return false;
	}

	fCarga.getline(tam, 4);

	anchura = (int)tam[0] - 48;
	altura = (int)tam[1] - 48;
	dificultad = (int)tam[2] - 48;

	fCarga.close();
	return true;
}

void cargar(int anchura, int altura, float*  tablero, char* fichero)
{
	char* array = (char*)malloc(anchura*altura + 1 + 3);
	std::ifstream fCarga(fichero);
	fCarga.getline(array, (anchura*altura + 1 + 3));
	for (int i = 0; i < anchura*altura; i++)
	{
		tablero[i] = array[i + 3] - 48;
	}
	free(array);
	fCarga.close();
}

void guardado(float* tablero, int anchura, int altura, int dificultad, char* fichero)
{
	//Sistema de guardado
	std::ofstream ficheroGuardado;
	ficheroGuardado.open(fichero);
	ficheroGuardado.clear();
	/* Almacenar anchura y altura*/
	ficheroGuardado << anchura;
	ficheroGuardado << altura;
	ficheroGuardado << dificultad;
	/* Almacenar Resto */
	for (int index = 0; index < anchura*altura; index++)
	{
		ficheroGuardado << tablero[index];
	}
	ficheroGuardado.close();
}

void bombaFila(float* tablero, int anchura, int altura, int dificultad, int fila) {

	for (int iFila = 0; (iFila + fila) < altura; iFila++)
	{
		for (int iColm = 0; iColm < anchura; iColm++)
		{
			if ((iFila + fila + 1) < altura)
			{
				tablero[(iFila + fila)*anchura + iColm] = tablero[(iFila + fila + 1)*altura + iColm];
			}
			else {
				tablero[(iFila + fila)*anchura + iColm] = generarJewel(dificultad);
			}
		}
	}
}

void bombaColumna(float* tablero, int anchura, int altura, int dificultad, int columna) {

	for (int iFila = 0; iFila < altura; iFila++)
	{
		for (int iColm = 0; (columna - iColm) > 0; iColm++)
		{
			if ((columna - iColm - 1) < 0)
			{
				tablero[(iFila*anchura) + (columna - iColm)] = generarJewel(dificultad);
			}
			else {
				tablero[(iFila*anchura) + (columna - iColm)] = tablero[(iFila*altura) + (columna - iColm - 1)];
			}
		}
	}
}

void bombaRotarCPU(float* tablero, int anchura, int altura, int fila, int columna)
{
	float aux[9];
	int index = 0;
	for (int iColm = columna - 1; iColm <= columna + 1; iColm++)
	{
		for (int iFila = fila + 1; iFila >= fila - 1; iFila--)
		{
			aux[index] = tablero[iFila*anchura + iColm];
			index++;
		}
	}
	index = 0;
	for (int iFila = 0; iFila < 3; iFila++)
	{
		for (int iColumna = 0; iColumna < 3; iColumna++)
		{
			tablero[(iFila + fila - 1)*anchura + (columna - 1) + iColumna] = aux[index];
			index++;
		}
	}
}

int main(int argc, char** argv) {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura;
	int altura;
	int dificultad;
	char modo;
	bool automatico = true;
	int TILE_WIDTH = 16;
	int size;
	char ficheroGuardado[9] = "save.txt";
	bool encontrado = false;
	int seleccion;

	float *tablero;
	/* Valores por argumento*/
	if (argc == 1)
	{
		std::cout << "Anchura del tablero: ";
		std::cin >> anchura;

		std::cout << "Altura del tablero: ";
		std::cin >> altura;

		std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil\n";
		std::cin >> dificultad;

		int seleccion;
		std::cout << "Automatico?   1.-SI   2.-NO\n";
		std::cin >> seleccion;
	}
	else
	{
		modo = argv[1][1];
		dificultad = atoi(argv[2]);
		anchura = atoi(argv[3]);
		altura = atoi(argv[4]);

		switch (modo) {
		case 'a': {seleccion = 1; break; }
		case 'm': {seleccion = 2; break; }
		default: printf("Valor no valido.\n"); return -1;
		}
	}
	
	bool jugando = true;

	/* Establecer automatico como modo de juego */
	
	size = anchura*altura;
	tablero = (float*)malloc(size * sizeof(float));

	//Se inicializa la matriz
	generacionInicialRandomJewels(tablero, dificultad, anchura, altura);

	//Bucle principal del juego
	while (jugando) {

		printTablero(tablero, anchura, altura);

		int jewel1_x = 0;
		int jewel1_y = 0;
		int accion = 0;

		std::cout << "Acci�n a realizar:\n";
		std::cout << "(1) Intercambiar Jewels\n";
		std::cout << "(2) Guardar partida\n";
		std::cout << "(3) Cargar partida\n";
		std::cout << "(9) Usar una Bomba\n";
		std::cout << "(0) Exit\n";
		std::cout << "Elija accion: ";

		std::cin >> accion;

		switch (accion) {
		case 0: {
			free(tablero);
			return 0;
		}
		case 1: {

			std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
			std::cout << "X: ";
			std::cin >> jewel1_x;
			std::cout << "Y: ";
			std::cin >> jewel1_y;

			if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
				printf("Posicion erronea.\n");
				continue;
			}

			int direccion = 0;
			std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
			std::cin >> direccion;

			if (direccion > 4 && direccion > 1) {
				printf("Direccion erronea.\n");
				continue;
			}
			else {
				switch (direccion)
				{
				case 1: //Arriba
				{
					if (jewel1_y == altura)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 2: //Abajo
				{
					if (jewel1_y == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 3: //Izquierda
				{
					if (jewel1_x == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 4: //Derecha
				{
					if (jewel1_x == anchura - 1)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				}

				intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura, seleccion, dificultad);

				if (seleccion == 1)
					analisisTableroAutomatico(dificultad, tablero, anchura, altura);
			}

			break;
		}
		case 2: {
			guardado(tablero, anchura, altura, dificultad, ficheroGuardado);
			std::cout << "Guardado correcto.\n";
			break;
		}
		case 3: {

			/* Precarga de tablero */
			encontrado = precargar(anchura, altura, dificultad, ficheroGuardado);

			if (encontrado)
			{
				/* Cargar tablero */
				cargar(anchura, altura, tablero, ficheroGuardado);
				std::cout << "Se ha cargado el Tablero: \n";
			}
			else {
				std::cout << "No existe ninguna partida guardada.\n";
			}
			break;

		}
		case 9: {
			// Bomba
			int bomba = 0;
			int fila = 0; int columna = 0;
			std::cout << "Elija una bomba:";

			/* Bombas por tipo de dificultad */
			switch (dificultad) {
			case 1: {
				std::cout << "(1) Bomba de fila ";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				std::cout << "X: ";
				std::cin >> fila;
				bombaFila(tablero, anchura, altura, dificultad, fila);
				break;
			}
			case 2: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				}
				break;
			}
			case 3: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "(3) Bomba de rotacion 3x3";
				std::cout << "\nEleccion: ";
				std::cin >> bomba;

				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				switch (bomba) {
				case 1:
				{
					std::cout << "X: ";
					std::cin >> fila;
					bombaFila(tablero, anchura, altura, dificultad, fila);
					break;
				}
				case 2:
				{
					std::cout << "Y: ";
					std::cin >> columna;
					bombaColumna(tablero, anchura, altura, dificultad, columna);
					break;
				}
				case 3:
				{
					for (int fila = 1; fila < anchura; fila += 3)
					{
						for (int columna = 1; columna < altura; columna += 3)
						{
							if ((fila - 1) < 0 || (fila + 1) >= altura || (columna - 1) < 0 || (columna + 1) >= anchura)
							{
								/* Se entra cuando no se puede rotar */
							}
							else
							{
								bombaRotarCPU(tablero, anchura, altura, fila, columna);
							}
						}
					}
					break;
				}
				}

				break;
			}
			}
			break;
		}

		}

	}
	free(tablero);
	return 0;
}