#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

//Funci�n que llamara a la de CUDA para actualizar la matrz
//void deleteJewels(float *A, int width) {
	//int size = width*width * sizeof(float);
	//float *A_d, *B_d, *C_d;

	//A y B a memoria GPU
	/*hipMalloc((void**)&A_d, size);
	hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
	hipMalloc((void**)&B_d, size);
	hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

	//Malloc en GPU de C
	hipMalloc((void**)&C_d, size);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(width, width);
	dim3 dimGrid(1, 1);

	//Inicio del calculo
	//Kernel << <dimGrid, dimBlock >> >(A_d, B_d, C_d, width);

	//Transfiere la solucion de la GPU al host
	hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

	//Libera memoria
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);*/
//}

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}

void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j+i*anchura]);
		}
	}
	printf("\n");
}

__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d,int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;
	//printf("\nJewels a eliminar horizontal: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);

	for (int i = 0; i < max; i++) {
		if ((tx == jewels_eliminadas_d[i]) && (ty * anchura) >(jewels_eliminadas_d[i + 1])) {
			tablero_d[tx + (ty - 1)*(anchura)] = tablero_d[tx + ty*anchura];
		}

		if (ty == altura) {
			//Generar jewel random
			hiprandState state;

			hiprand_init((unsigned long long)clock(), i, 0, &state);

			tablero_d[tx + ty*anchura] = hiprand_uniform(&state);
		}
		i++;
	}
}

//Elimina las jewels recibidas, bajas las filas para rellenas, y genera arriba del todo jewels nuevas. TODO
void eliminarJewels(float* tablero, float* jewels_eliminadas,int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	eliminarJewelsKernel <<<dimGrid, dimBlock >>>(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}

__global__ void analisisTableroKernel(float *tablero_d, float *jewels_eliminadas_d, int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//printf("\ntx:%i ty:%i\n",tx,ty);

	if (tx > 1 && tx < anchura - 1) {
		if (tablero_d[tx + anchura*ty] == tablero_d[tx + 1 + anchura*ty] && tablero_d[tx + anchura*ty] == tablero_d[tx - 1 + anchura*ty]){
			jewels_eliminadas_d[0] = tx - 1;
			jewels_eliminadas_d[1] = anchura*ty;
			jewels_eliminadas_d[2] = tx;
			jewels_eliminadas_d[3] = anchura*ty;
			jewels_eliminadas_d[4] = tx + 1;
			jewels_eliminadas_d[5] = anchura*ty;
			//printf("\nJewels a eliminar horizontal: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1]/anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);
		}
	}

	if (ty > 1 && ty < altura - 1) {
		if (tablero_d[tx + anchura*ty] == tablero_d[tx + anchura*(ty + 1)] && tablero_d[tx + anchura*ty] == tablero_d[tx + anchura*(ty - 1)]) {
			jewels_eliminadas_d[0] = tx;
			jewels_eliminadas_d[1] = anchura*(ty - 1);
			jewels_eliminadas_d[2] = tx;
			jewels_eliminadas_d[3] = anchura*ty;
			jewels_eliminadas_d[4] = tx;
			jewels_eliminadas_d[5] = anchura*(ty + 1);
			//printf("\nty: %i\n",ty);
			//printf("\nJewels a eliminar vertical: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], (jewels_eliminadas_d[1]/ anchura), jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], (jewels_eliminadas_d[5] / anchura));
		}
	}
}

//Funcion CPU
void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tiene posibles eliminables por la derecha o izquierda
	if (((x-1+y*anchura>=0)&&tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) || ((x+1+y*anchura<=size)&&tablero[x + 1 + y*anchura] == tablero[x + y*anchura])) {
		int jewels_posibles_izq = 0;
		int jewels_posibles_der = 0;

		//Si tiene por la izquierda
		if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x - i + y*anchura >= 0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_izq++;
				i++;
			}
		}

		//Si tiene por la derecha
		if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x + i+ y*anchura <= size) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_der++;
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
			int salto = 0;

			for (int j = 0; j <= (jewels_posibles_izq + 1);j++) {
				jewels_eliminadas[salto]=x-j;
				jewels_eliminadas[salto + 1]=y;
				salto += 2;
			}

			salto = 0;
			for (int k = 1; k <= jewels_posibles_der; k++) {
				jewels_eliminadas[salto + 1 + jewels_posibles_izq] = x + k;
				jewels_eliminadas[salto + 1 + jewels_posibles_izq+1] = y;
				salto += 2;
			}
		}
	} else {
		int jewels_posibles_arrib = 0;
		int jewels_posibles_abaj = 0;

		//Si tiene por abajo
		if ((x + (y-1)*anchura >= 0) && tablero[x + (y-1)*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x + (y-i)*anchura >= 0) && tablero[x + (y-i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_abaj++;
				i++;
			}
		}

		//Si tiene por arriba
		if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x + (y+i)*anchura <= size) && tablero[x + (y+i)*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_arrib++;
				i++;
			}
		}

		//Se pueden eliminar
		if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {
			int salto = 0;

			for (int j = 0; j <= (jewels_posibles_abaj + 1); j++) {
				jewels_eliminadas[salto] = x;
				jewels_eliminadas[salto + 1] = y - j;
				salto += 2;
			}

			salto = 0;
			for (int k = 1; k <= jewels_posibles_arrib; k++) {
				jewels_eliminadas[salto + 1 + jewels_posibles_abaj] = x;
				jewels_eliminadas[salto + 1 + jewels_posibles_abaj + 1] = y + k;
				salto += 2;
			}
		}
	}

	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
}

//CUDA CPU Function
/*void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	analisisTableroKernel <<<dimGrid, dimBlock>>>(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);
	printf("\nSali!\n");

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(jewels_eliminadas, jewels_eliminadas_d, max*sizeof(float), hipMemcpyDeviceToHost);

	printTablero(tablero, anchura, altura);
	printf("Pulse una tecla para continuar...");
	int relleno = 0;
	std::cin >> relleno;
	if (jewels_eliminadas[0] == -1 && jewels_eliminadas[1]==-1) {
		//Se eliminan las jewels seleccionadas, se bajan las superiores y se generan nuevas
		hipFree(tablero_d);
		hipFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		analisisTableroManual(dificultad, tablero, anchura, altura);
	}
	else {
		hipFree(tablero_d);
		hipFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
		printTablero(tablero, anchura, altura);
	}
}*/

//CUDA CPU Function. Analiza la mejor opcion y la ejecuta
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	analisisTableroKernel << <dimGrid, dimBlock >> >(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);
	printf("\nSali!\n");

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(jewels_eliminadas, jewels_eliminadas_d, max * sizeof(float), hipMemcpyDeviceToHost);

	printTablero(tablero, anchura, altura);
	printf("Pulse una tecla para continuar...");
	int relleno = 0;
	std::cin >> relleno;
	if (jewels_eliminadas[0] == -1 && jewels_eliminadas[1] == -1) {
		//Se eliminan las jewels seleccionadas, se bajan las superiores y se generan nuevas
		hipFree(tablero_d);
		hipFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		analisisTableroAutomatico(dificultad, tablero, anchura, altura);
	}
	else {
		hipFree(tablero_d);
		hipFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
		printTablero(tablero, anchura, altura);
	}
}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion) 
	{
		case 1: //Arriba
		{
			jewel2_y += 1;
			break;
		}
		case 2: //Abajo
		{
			jewel2_y -= 1;
			break;
		}
		case 3: //Izquierda
		{
			jewel2_x -= 1;
			break;
		}
		case 4: //Derecha
		{
			jewel2_x += 1;
			break;
		}
	}
	int aux1;

	aux1 = tablero[jewel2_x+jewel2_y*anchura];

	tablero[jewel2_x+jewel2_y*anchura] = tablero[jewel1_x+jewel1_y*anchura];
	tablero[jewel1_x+jewel1_y*anchura] = aux1;
}

int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int TILE_WIDTH = 16;

	float *tablero;
	bool jugando = true;

	std::cout << "Anchura del tablero: ";
	std::cin >> anchura;

	std::cout << "Altura del tablero: ";
	std::cin >> altura;

	std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil";
	std::cin >> dificultad;

	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO";
	std::cin >> seleccion;

	switch (seleccion) {
		case 1: automatico = true; break;
		case 2: automatico = false; break;
		default: printf("Valor no valido.\n"); return -1;
	}

	tablero = (float*)malloc(altura * anchura * sizeof(float));

	//Se inicializa la matriz
	generacionInicialRandomJewels(tablero, dificultad, anchura, altura);

	//Bucle principal del juego
	while (jugando) {

		if (seleccion == 2)
			analisisTableroManual(dificultad, tablero, anchura, altura);
		else
			if (seleccion == 1)
				analisisTableroAutomatico();

		int jewel1_x = 0;
		int jewel1_y = 0;
		int accion = 0;

		std::cout << "Acci�n a realizar:";
		std::cout << "(1) Intercambiar Jewels";
		std::cout << "(2) Usar una Bomba";

		std::cin >> accion;

		if(accion == 1){
			std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
			std::cout << "X: ";
			std::cin >> jewel1_x;
			std::cout << "Y: ";
			std::cin >> jewel1_y;

			if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
				printf("Posicion erronea.\n");
				continue;
			}

			int direccion = 0;
			std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
			std::cin >> direccion;

			if (direccion > 4 && direccion > 1) {
				printf("Direccion erronea.\n");
				continue;
			}
			else {
				switch (direccion)
				{
				case 1: //Arriba
				{
					if (jewel1_y == altura)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 2: //Abajo
				{
					if (jewel1_y == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 3: //Izquierda
				{
					if (jewel1_x == 0)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				case 4: //Derecha
				{
					if (jewel1_x == anchura - 1)
					{
						printf("No se puede realizar el intercambio especificado.\n");
						continue;
					}
					break;
				}
				}

				intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura);
			}

		}else {
			int bomba = 0;

			std::cout << "Elija una bomba:";

			switch (dificultad) {
				case 1: {
					std::cout << "(1) Bomba de fila";
					break;
				}
				case 2: {
					std::cout << "(1) Bomba de fila";
					std::cout << "(2) Bomba de columna";
					break;
				}
				case 3: {
					std::cout << "(1) Bomba de fila";
					std::cout << "(2) Bomba de columna";
					std::cout << "(3) Bomba de rotacion 3x3 (la jewel elegida es el centro)";
					break;
				}
			}

			std::cin >> bomba;

			switch (dificultad)
			{
			case 1:
			{
				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				break;
			}
			case 2:
			{
				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				break;
			}
			case 3:
			{
				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				break;
			}
			}

			//LLAMADA A LA FUNCION DE EJECUTAR BOMBA//
		}
	}
	return 0;
}