#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <fstream>

//Funci�n que llamara a la de CUDA para actualizar la matrz
//void deleteJewels(float *A, int width) {
	//int size = width*width * sizeof(float);
	//float *A_d, *B_d, *C_d;

	//A y B a memoria GPU
	/*cudaMalloc((void**)&A_d, size);
	cudaMemcpy(A_d, A, size, cudaMemcpyHostToDevice);
	cudaMalloc((void**)&B_d, size);
	cudaMemcpy(B_d, B, size, cudaMemcpyHostToDevice);

	//Malloc en GPU de C
	cudaMalloc((void**)&C_d, size);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(width, width);
	dim3 dimGrid(1, 1);

	//Inicio del calculo
	//Kernel << <dimGrid, dimBlock >> >(A_d, B_d, C_d, width);

	//Transfiere la solucion de la GPU al host
	cudaMemcpy(C, C_d, size, cudaMemcpyDeviceToHost);

	//Libera memoria
	cudaFree(A_d);
	cudaFree(B_d);
	cudaFree(C_d);*/
//}

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
	return -1;
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}

void printTablero(float* tablero, int anchura, int altura) {
	for (int i = altura - 1; i >= 0; i--) {
		printf("\n");
		for (int j = 0; j < anchura; j++) {
			printf("%d ", (int)tablero[j+i*anchura]);
		}
	}
	printf("\n");
}

//TODO: revisar
void eliminarJewelsCPU(float* tablero, float* jewels_eliminadas, int dificultad, int anchura, int altura) {
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;
	//printf("\nJewels a eliminar horizontal: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);
	for (int y = 0; y < altura; y++) {
		for (int x = 0; x < anchura; x++) {
			for (int i = 0; i < max; i++) {
				if ((x == jewels_eliminadas[i]) && (y * anchura) > (jewels_eliminadas[i + 1])) {
					tablero[x + (y - 1)*(anchura)] = tablero[x + y*anchura];
				}

				if (y == altura) {
					//Generar jewel random

					switch (dificultad) {
					case 1: {
						int randJewel = rand() % 4 + 1;
						tablero[x+y*anchura] = randJewel;
						break;
					}
					case 2: {
						int randJewel = rand() % 6 + 1;
						tablero[x+y*anchura] = randJewel;
						break;
					}
					case 3: {
						int randJewel = rand() % 8 + 1;
						tablero[x + y*anchura] = randJewel;
						break;
					}
					}
				}
				i++;
			}
		}
	}
}

//TODO: Usar tx y ty como doble for anidado
__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d,int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;
	printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);

	for (int i = 0; i < max; i++) {
		if ((tx == jewels_eliminadas_d[i]) && (ty)>(jewels_eliminadas_d[i + 1])) {
			tablero_d[tx + (ty - 1)*(anchura)] = tablero_d[tx + ty*anchura];
		}

		if (ty == altura) {
			//Generar jewel random
			hiprandState state;

			hiprand_init((unsigned long long)clock(), i, 0, &state);

			tablero_d[tx + ty*anchura] = hiprand_uniform(&state);
		}
		i++;
	}
}

//Elimina las jewels recibidas, bajas las filas para rellenas, y genera arriba del todo jewels nuevas. TODO
void eliminarJewels(float* tablero, float* jewels_eliminadas,int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	eliminarJewelsKernel <<<dimGrid, dimBlock >>>(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(tablero, tablero_d, size, hipMemcpyDeviceToHost);

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}

/*__global__ void analisisTableroKernel(float *tablero_d, float *jewels_eliminadas_d, int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	//printf("\ntx:%i ty:%i\n",tx,ty);

	if (tx > 1 && tx < anchura - 1) {
		if (tablero_d[tx + anchura*ty] == tablero_d[tx + 1 + anchura*ty] && tablero_d[tx + anchura*ty] == tablero_d[tx - 1 + anchura*ty]){
			jewels_eliminadas_d[0] = tx - 1;
			jewels_eliminadas_d[1] = anchura*ty;
			jewels_eliminadas_d[2] = tx;
			jewels_eliminadas_d[3] = anchura*ty;
			jewels_eliminadas_d[4] = tx + 1;
			jewels_eliminadas_d[5] = anchura*ty;
			//printf("\nJewels a eliminar horizontal: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1]/anchura, jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], jewels_eliminadas_d[5] / anchura);
		}
	}

	if (ty > 1 && ty < altura - 1) {
		if (tablero_d[tx + anchura*ty] == tablero_d[tx + anchura*(ty + 1)] && tablero_d[tx + anchura*ty] == tablero_d[tx + anchura*(ty - 1)]) {
			jewels_eliminadas_d[0] = tx;
			jewels_eliminadas_d[1] = anchura*(ty - 1);
			jewels_eliminadas_d[2] = tx;
			jewels_eliminadas_d[3] = anchura*ty;
			jewels_eliminadas_d[4] = tx;
			jewels_eliminadas_d[5] = anchura*(ty + 1);
			//printf("\nty: %i\n",ty);
			//printf("\nJewels a eliminar vertical: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], (jewels_eliminadas_d[1]/ anchura), jewels_eliminadas_d[2], jewels_eliminadas_d[3] / anchura, jewels_eliminadas_d[4], (jewels_eliminadas_d[5] / anchura));
		}
	}
}*/

//Funcion CPU. FUNCIONA HORIZONTAL, FALTA VERTICAL CORREGIR
void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura, int x, int y) {
	int max = 0;
	int size = anchura*altura;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan MAX jewels como mucho, se guardan sus x e y
	float* jewels_eliminadas = (float*)malloc(2 * max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tiene posibles eliminables por la derecha o izquierda
	// (((x-1+y*anchura>=0)&&tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) || ((x+1+y*anchura<=size)&&tablero[x + 1 + y*anchura] == tablero[x + y*anchura])) {
		int jewels_posibles_izq = 0;
		int jewels_posibles_der = 0;
		printf("\nHORIZONTAL\n");
		//Si tiene por la izquierda
		if ((x - 1 + y*anchura >= 0) && tablero[x - 1 + y*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x - i + y*anchura >= 0) && tablero[x - i + y*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_izq++;
				i++;
			}
		}

		//Si tiene por la derecha
		if ((x + 1 + y*anchura <= size) && tablero[x + 1 + y*anchura] == tablero[x + y*anchura]) {
			int i = 1;
			while ((x + i+ y*anchura <= size) && tablero[x + i + y*anchura] == tablero[x + y*anchura]) {
				jewels_posibles_der++;
				i++;
			}
		}

		//Se pueden eliminar horizontalmente
		if (1 + jewels_posibles_izq + jewels_posibles_der >= 3) {
			jewels_eliminadas[0] = x;
			jewels_eliminadas[1] = y;

			int salto = 2;

			for (int j = 1; j <= (jewels_posibles_izq);j++) {
				jewels_eliminadas[salto]=x-j;
				jewels_eliminadas[salto + 1]=y;
				salto += 2;
			}

			salto = 2;
			for (int k = 1; k <= jewels_posibles_der; k++) {
				jewels_eliminadas[salto + 1 + jewels_posibles_izq] = x + k;
				jewels_eliminadas[salto + 1 + jewels_posibles_izq+1] = y;
				salto += 2;
			}
		} else {	//Analizamos la vertical
			int jewels_posibles_arrib = 0;
			int jewels_posibles_abaj = 0;

			printf("\nVERTICAL\n");
			//Si tiene por abajo
			if ((x + (y - 1)*anchura >= 0) && tablero[x + (y - 1)*anchura] == tablero[x + y*anchura]) {
				printf("\nABAJO\n");
				int i = 1;
				while ((x + (y - i)*anchura >= 0) && tablero[x + (y - i)*anchura] == tablero[x + y*anchura]) {
					jewels_posibles_abaj++;
					printf("\nTIENE ABAJO\n");
					i++;
				}
			}

			//Si tiene por arriba
			if ((x + 1 + y*anchura <= size) && tablero[x + (y + 1)*anchura] == tablero[x + y*anchura]) {
				printf("\nARRIBA\n");
				int i = 1;
				while ((x + (y + i)*anchura <= size) && tablero[x + (y + i)*anchura] == tablero[x + y*anchura]) {
					jewels_posibles_arrib++;
					printf("\nTIENE ARRIBA\n");
					i++;
				}
			}

			//Se pueden eliminar
			if (1 + jewels_posibles_abaj + jewels_posibles_arrib >= 3) {
				printf("\nSE PUEDE\n");

				jewels_eliminadas[0] = x;
				jewels_eliminadas[1] = y;

				int salto = 2;
				for (int j = 1; j <= (jewels_posibles_abaj); j++) {
					jewels_eliminadas[salto] = x;
					jewels_eliminadas[salto + 1] = y - j;
					salto += 2;
				}

				salto = 2;
				for (int k = 1; k <= jewels_posibles_arrib; k++) {
					jewels_eliminadas[salto + jewels_posibles_abaj] = x;
					jewels_eliminadas[salto + 1 + jewels_posibles_abaj + 1] = y + k;
					salto += 2;
				}
			}
		}
		
	//("\nJewels a eliminar horizontal: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas_d[0], jewels_eliminadas_d[1] / anchura, jewels_eliminadas_d[2], jewels_eliminadas_3] / anchura, jewels_eliminadas[4], jewels_eliminadas[5] / anchura);
	for (int q = 0; q < 2*max; q++) {
		if (q % 2 != 0) {
			printf(" y:%f\n",jewels_eliminadas[q]);
		}
		else {
			printf("| x:%f\n", jewels_eliminadas[q]);
		}
	}
	if(jewels_eliminadas[0]!=-1)
		eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
}

//CUDA CPU Function
/*void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tablero a GPU
	cudaMalloc((void**)&tablero_d, size);
	cudaMemcpy(tablero_d, tablero, size, cudaMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	cudaMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	cudaMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), cudaMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	analisisTableroKernel <<<dimGrid, dimBlock>>>(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);
	printf("\nSali!\n");

	//Transfiere las jewels a eliminar de la GPU al host
	cudaMemcpy(jewels_eliminadas, jewels_eliminadas_d, max*sizeof(float), cudaMemcpyDeviceToHost);

	printTablero(tablero, anchura, altura);
	printf("Pulse una tecla para continuar...");
	int relleno = 0;
	std::cin >> relleno;
	if (jewels_eliminadas[0] == -1 && jewels_eliminadas[1]==-1) {
		//Se eliminan las jewels seleccionadas, se bajan las superiores y se generan nuevas
		cudaFree(tablero_d);
		cudaFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		analisisTableroManual(dificultad, tablero, anchura, altura);
	}
	else {
		cudaFree(tablero_d);
		cudaFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
		printTablero(tablero, anchura, altura);
	}
}*/

//CUDA CPU Function. Analiza la mejor opcion y la ejecuta
void analisisTableroAutomatico(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	//analisisTableroKernel << <dimGrid, dimBlock >> >(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);
	//printf("\nSali!\n");

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(jewels_eliminadas, jewels_eliminadas_d, max * sizeof(float), hipMemcpyDeviceToHost);

	if (jewels_eliminadas[0] == -1 && jewels_eliminadas[1] == -1) {
		//Se eliminan las jewels seleccionadas, se bajan las superiores y se generan nuevas
		hipFree(tablero_d);
		hipFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		analisisTableroAutomatico(dificultad, tablero, anchura, altura);
	}
	else {
		hipFree(tablero_d);
		hipFree(jewels_eliminadas_d);

		//printf("\nJewels a eliminar: x:%f y:%f | x:%f y:%f | x:%f y:%f", jewels_eliminadas[0], jewels_eliminadas[1]/anchura, jewels_eliminadas[2], jewels_eliminadas[3]/anchura, jewels_eliminadas[4], jewels_eliminadas[5]/anchura);
		eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);
		printTablero(tablero, anchura, altura);
	}
}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura, int seleccion,int dificultad) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion) 
	{
		case 1: //Arriba
		{
			jewel2_y += 1;
			break;
		}
		case 2: //Abajo
		{
			jewel2_y -= 1;
			break;
		}
		case 3: //Izquierda
		{
			jewel2_x -= 1;
			break;
		}
		case 4: //Derecha
		{
			jewel2_x += 1;
			break;
		}
	}
	int aux1;

	aux1 = tablero[jewel2_x+jewel2_y*anchura];

	tablero[jewel2_x+jewel2_y*anchura] = tablero[jewel1_x+jewel1_y*anchura];
	tablero[jewel1_x+jewel1_y*anchura] = aux1;

	if (seleccion == 2)
		analisisTableroManual(dificultad, tablero, anchura, altura, jewel2_x, jewel2_y);
}
bool precargar(int& anchura, int& altura, int& dificultad, char* fichero)
{
	std::ifstream fCarga(fichero);
	char tam[4];
	if (!fCarga.is_open())
	{
		std::cout << "ERROR: no existe un archivo guardado." << std::endl;
		return false;
	}
	
	fCarga.getline(tam, 4);

	anchura = (int)tam[0] - 48;
	altura = (int)tam[1] - 48;
	dificultad = (int)tam[2] -48;

	fCarga.close();
	return true;
}
void cargar(int anchura, int altura, float*  tablero, char* fichero)
{
	char* array = (char*)malloc(anchura*altura + 1 + 3);
	std::ifstream fCarga(fichero);
	fCarga.getline(array, (anchura*altura + 1 + 3));
	for (int i = 0; i < anchura*altura; i++)
	{
		tablero[i] = array[i + 3];
	}
	free(array);
	fCarga.close();
}

void guardado(float* tablero, int anchura, int altura, int dificultad, char* fichero)
{
	//Sistema de guardado
	std::ofstream ficheroGuardado;
	ficheroGuardado.open(fichero);
	ficheroGuardado.clear();
	/* Almacenar anchura y altura*/
	ficheroGuardado << anchura;
	ficheroGuardado << altura;
	ficheroGuardado << dificultad;
	/* Almacenar Resto */
	for (int index = 0; index < anchura*altura; index++)
	{
		ficheroGuardado << tablero[index];
	}
	ficheroGuardado.close();
}

int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int TILE_WIDTH = 16;
	char ficheroGuardado[9] = "save.txt";

	float *tablero;
	bool jugando = true;

	int eleccion;
	bool encontrado = false;
	std::cout << "Desea cargar una partida guardada? 1.-SI   2.-NO\n";
	std::cin >> eleccion;
	if (eleccion == 1)
	{
		encontrado = precargar(anchura, altura, dificultad, ficheroGuardado);
	}

	if (!encontrado || (eleccion == 2))
	{
		std::cout << "Anchura del tablero: ";
		std::cin >> anchura;

		std::cout << "Altura del tablero: ";
		std::cin >> altura;

		std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil\n";
		std::cin >> dificultad;
	}
	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO\n";
	std::cin >> seleccion;

	switch (seleccion) {
		case 1: automatico = true; break;
		case 2: automatico = false; break;
		default: printf("Valor no valido.\n"); return -1;
	}
	
	tablero = (float*)malloc(altura * anchura * sizeof(float));

	//Se inicializa la matriz
	if (encontrado)
	{
		cargar(anchura, altura, tablero, ficheroGuardado);
	}
	generacionInicialRandomJewels(tablero, dificultad, anchura, altura);
	
	//Bucle principal del juego
	while (jugando) {

		printTablero(tablero, anchura, altura);
		
		int jewel1_x = 0;
		int jewel1_y = 0;
		int accion = 0;

		std::cout << "Acci�n a realizar:\n";
		std::cout << "(1) Intercambiar Jewels\n";
		std::cout << "(2) Usar una Bomba\n";
		std::cout << "(3) Guardar partida\n";
		std::cout << "Elija accion: ";

		std::cin >> accion;

		
		switch (accion) {
		case 1: {
			
				std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
				std::cout << "X: ";
				std::cin >> jewel1_x;
				std::cout << "Y: ";
				std::cin >> jewel1_y;

				if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
					printf("Posicion erronea.\n");
					continue;
				}

				int direccion = 0;
				std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
				std::cin >> direccion;

				if (direccion > 4 && direccion > 1) {
					printf("Direccion erronea.\n");
					continue;
				}
				else {
					switch (direccion)
					{
					case 1: //Arriba
					{
						if (jewel1_y == altura)
						{
							printf("No se puede realizar el intercambio especificado.\n");
							continue;
						}
						break;
					}
					case 2: //Abajo
					{
						if (jewel1_y == 0)
						{
							printf("No se puede realizar el intercambio especificado.\n");
							continue;
						}
						break;
					}
					case 3: //Izquierda
					{
						if (jewel1_x == 0)
						{
							printf("No se puede realizar el intercambio especificado.\n");
							continue;
						}
						break;
					}
					case 4: //Derecha
					{
						if (jewel1_x == anchura - 1)
						{
							printf("No se puede realizar el intercambio especificado.\n");
							continue;
						}
						break;
					}
					}

					intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura, seleccion, dificultad);

					if (seleccion == 1)
						analisisTableroAutomatico(dificultad, tablero, anchura, altura);
				}

			break;
		}
		case 2: {
			// Bomba
			int bomba = 0;

			std::cout << "Elija una bomba:";

			switch (dificultad) {
			case 1: {
				std::cout << "(1) Bomba de fila";
				break;
			}
			case 2: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				break;
			}
			case 3: {
				std::cout << "(1) Bomba de fila";
				std::cout << "(2) Bomba de columna";
				std::cout << "(3) Bomba de rotacion 3x3 (la jewel elegida es el centro)";
				break;
			}
			}

			std::cin >> bomba;

			switch (dificultad)
			{
			case 1:
			{
				if (bomba != 1)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				break;
			}
			case 2:
			{
				if (bomba < 1 && bomba > 2)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				break;
			}
			case 3:
			{
				if (bomba < 1 && bomba > 3)
				{
					printf("Bomba erronea.\n");
					continue;
				}
				break;
			}
			}

			//LLAMADA A LA FUNCION DE EJECUTAR BOMBA//


			break;
		}
		case 3: {
			guardado(tablero, anchura, altura, dificultad, ficheroGuardado);
			std::cout << "Guardado correcto.\n";
			break;
		}
		}
			
	}
	return 0;
}