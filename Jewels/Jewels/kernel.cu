#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

//Funci�n que llamara a la de CUDA para actualizar la matrz
void deleteJewels(float *A, int width) {
	//int size = width*width * sizeof(float);
	//float *A_d, *B_d, *C_d;

	//A y B a memoria GPU
	/*cudaMalloc((void**)&A_d, size);
	cudaMemcpy(A_d, A, size, cudaMemcpyHostToDevice);
	cudaMalloc((void**)&B_d, size);
	cudaMemcpy(B_d, B, size, cudaMemcpyHostToDevice);

	//Malloc en GPU de C
	cudaMalloc((void**)&C_d, size);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(width, width);
	dim3 dimGrid(1, 1);

	//Inicio del calculo
	//Kernel << <dimGrid, dimBlock >> >(A_d, B_d, C_d, width);

	//Transfiere la solucion de la GPU al host
	cudaMemcpy(C, C_d, size, cudaMemcpyDeviceToHost);

	//Libera memoria
	cudaFree(A_d);
	cudaFree(B_d);
	cudaFree(C_d);*/
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int altura, int anchura) {
	for (int i = 0; i < altura*anchura; i++) {

	}
}

int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	int TILE_WIDTH = 16;

	float *tablero;
	bool jugando = true;

	std::cout << "Altura del tablero: ";
	std::cin >> altura;

	std::cout << "Anchura del tablero: ";
	std::cin >> anchura;

	std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil";
	std::cin >> dificultad;

	tablero = (float*)malloc(altura * anchura * sizeof(float));

	//Se inicializa la matriz
	generacionInicialRandomJewels(tablero, dificultad, altura, anchura);


	//Bucle principal del juego
	while (jugando) {

	}

	/*const int width = 3;
	float A[width*width], B[width*width], C[width*width];
	for (int i = 0; i < (width*width); i++) {
		int valor = 0;
		std::cout << "Valor en A (de izquierda a derecha, por filas): ";
		std::cin >> valor;
		A[i] = valor;
		valor = 0;
		std::cout << "Valor en B (de izquierda a derecha, por filas): ";
		std::cin >> valor;
		B[i] = valor;
		C[i] = 0;
	}
	MatrixMultiplication(A, B, C, width);
	printf("Solucion: \n");
	for (int i = 0; i < (width*width); i++) {
		if (i%width == 0) { printf("\n"); }
		printf("%f ", C[i]);
	}

	int exit;
	scanf("%d", &exit);*/

	return 0;
}