#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

//Funci�n que llamara a la de CUDA para actualizar la matrz
//void deleteJewels(float *A, int width) {
	//int size = width*width * sizeof(float);
	//float *A_d, *B_d, *C_d;

	//A y B a memoria GPU
	/*cudaMalloc((void**)&A_d, size);
	cudaMemcpy(A_d, A, size, cudaMemcpyHostToDevice);
	cudaMalloc((void**)&B_d, size);
	cudaMemcpy(B_d, B, size, cudaMemcpyHostToDevice);

	//Malloc en GPU de C
	cudaMalloc((void**)&C_d, size);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(width, width);
	dim3 dimGrid(1, 1);

	//Inicio del calculo
	//Kernel << <dimGrid, dimBlock >> >(A_d, B_d, C_d, width);

	//Transfiere la solucion de la GPU al host
	cudaMemcpy(C, C_d, size, cudaMemcpyDeviceToHost);

	//Libera memoria
	cudaFree(A_d);
	cudaFree(B_d);
	cudaFree(C_d);*/
//}

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
		break;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
		break;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
		break;
	}
	}
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}

void printTablero(float* tablero, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		if (i%anchura == 0)
			printf("\n");
		printf("%d ",(int)tablero[i]);
	}
}

//Elimina las jewels recibidas, bajas las filas para rellenas, y genera arriba del todo jewels nuevas. TODO
void eliminarJewels() {

}

//CUDA CPU Function
void analisisTableroManual(int dificultad, float* tablero[], int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(6 * sizeof(float));

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, 6 * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, 6 * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	//analisisTableroKernel << <dimGrid, dimBlock >> >(tablero_d, jewels_eliminadas_d, anchura, altura);

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(jewels_eliminadas, jewels_eliminadas_d, size, hipMemcpyDeviceToHost);

	//Se eliminan las jewels seleccionadas, se bajan las superiores y se generan nuevas
	//eliminarJewels(tablero, jewels_eliminadas, anchura, altura);

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}

//CUDA CPU Function.
void analisisTableroAutomatico() {

}

//CUDA CPU Function. TODO
void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion) 
	{
		case 1: 
		{
			if (jewel1_y == 0)
			{
				jewel2_y = altura - 1;
			}
			else
			{
				jewel2_y = jewel1_x - 1;
			}
			break;
		}
		case 2:
		{
			if (jewel1_y == altura - 1 )
			{
				jewel2_y = 0;
			}
			else
			{
				jewel2_y = jewel1_y + 1;
			}
			break;
		}
		case 3:
		{
			if (jewel1_x == 0)
			{
				jewel2_x = anchura - 1;
			}
			else
			{
				jewel2_x = jewel1_x - 1;
			}
			break;
		}
		case 4:
		{
			if (jewel1_x == anchura - 1)
			{
				jewel2_x = 0;
			}
			else
			{
				jewel2_x = jewel1_x + 1;
			}
			break;
		}
	}

	/* CPU mode */
	int aux1;

	aux1 = tablero[(jewel1_x + 1)*(jewel1_y + 1)];

	tablero[(jewel1_x + 1)*(jewel1_y + 1)] = tablero[(jewel2_x + 1)*(jewel2_y + 1)];
	tablero[(jewel2_x + 1)*(jewel2_y + 1)] = aux1;
}

int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int TILE_WIDTH = 16;

	float *tablero;
	bool jugando = true;

	std::cout << "Anchura del tablero: ";
	std::cin >> anchura;

	std::cout << "Altura del tablero: ";
	std::cin >> altura;

	std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil";
	std::cin >> dificultad;

	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO";
	std::cin >> seleccion;

	switch (seleccion) {
	case 1: automatico = true; break;
	case 2: automatico = false; break;
	default: printf("Valor no valido.\n"); return -1;
	}

	tablero = (float*)malloc(altura * anchura * sizeof(float));

	//Se inicializa la matriz
	generacionInicialRandomJewels(tablero, dificultad, anchura, altura);

	//Bucle principal del juego
	while (jugando) {
		printTablero(tablero, anchura, altura);
		//analisisTableroManual();

		int jewel1_x = 0;
		int jewel1_y = 0;
		std::cout << "Posicion de la primera jewel (empiezan en 0)\n";
		std::cout << "X: ";
		std::cin >> jewel1_x;
		std::cout << "Y: ";
		std::cin >> jewel1_y;

		if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
			printf("Posicion erronea.\n");
			continue;
		}

		int direccion = 0;
		std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
		std::cin >> direccion;

		if (direccion > 4 && direccion > 1) {
			printf("Direccion erronea.\n");
			continue;
		}

		intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion,anchura,altura);

	}

	/*const int width = 3;
	float A[width*width], B[width*width], C[width*width];
	for (int i = 0; i < (width*width); i++) {
		int valor = 0;
		std::cout << "Valor en A (de izquierda a derecha, por filas): ";
		std::cin >> valor;
		A[i] = valor;
		valor = 0;
		std::cout << "Valor en B (de izquierda a derecha, por filas): ";
		std::cin >> valor;
		B[i] = valor;
		C[i] = 0;
	}
	MatrixMultiplication(A, B, C, width);
	printf("Solucion: \n");
	for (int i = 0; i < (width*width); i++) {
		if (i%width == 0) { printf("\n"); }
		printf("%f ", C[i]);
	}

	int exit;
	scanf("%d", &exit);*/

	return 0;
}