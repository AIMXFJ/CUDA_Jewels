#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

//Funci�n que llamara a la de CUDA para actualizar la matrz
//void deleteJewels(float *A, int width) {
	//int size = width*width * sizeof(float);
	//float *A_d, *B_d, *C_d;

	//A y B a memoria GPU
	/*cudaMalloc((void**)&A_d, size);
	cudaMemcpy(A_d, A, size, cudaMemcpyHostToDevice);
	cudaMalloc((void**)&B_d, size);
	cudaMemcpy(B_d, B, size, cudaMemcpyHostToDevice);

	//Malloc en GPU de C
	cudaMalloc((void**)&C_d, size);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(width, width);
	dim3 dimGrid(1, 1);

	//Inicio del calculo
	//Kernel << <dimGrid, dimBlock >> >(A_d, B_d, C_d, width);

	//Transfiere la solucion de la GPU al host
	cudaMemcpy(C, C_d, size, cudaMemcpyDeviceToHost);

	//Libera memoria
	cudaFree(A_d);
	cudaFree(B_d);
	cudaFree(C_d);*/
//}

//funcion para generar una jewel aleatoria, como la generacion inicial.
int generarJewel(int dificultad) {
	switch (dificultad) {
	case 1: {
		int randJewel = rand() % 4 + 1;
		return randJewel;
	}
	case 2: {
		int randJewel = rand() % 6 + 1;
		return randJewel;
	}
	case 3: {
		int randJewel = rand() % 8 + 1;
		return randJewel;
	}
	}
}

void generacionInicialRandomJewels(float *tablero, int dificultad, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		switch (dificultad) {
		case 1: {
			int randJewel = rand() % 4 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 2: {
			int randJewel = rand() % 6 + 1;
			tablero[i] = randJewel;
			break;
			}
		case 3: {
			int randJewel = rand() % 8 + 1;
			tablero[i] = randJewel;
			break;
		}
		}
	}
}

void printTablero(float* tablero, int anchura, int altura) {
	for (int i = 0; i < altura*anchura; i++) {
		if (i%anchura == 0)
			printf("\n");
		printf("%d ",(int)tablero[i]);
	}
}

__global__ void eliminarJewelsKernel(float* tablero_d, float* jewels_eliminadas_d,int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	for (int i = 0; i < max; i+2) {
		//La jewel esta en la columna de la que hay que eliminar, sin ser esta ultima
		if ((tx == jewels_eliminadas_d[i]) && (ty * altura) > (jewels_eliminadas_d[i+1])) {
			tablero_d[tx + ty*(altura - 1)] = tablero_d[tx + ty*altura];
		}

		if (ty == altura) {
			//Generar jewel random
			hiprandState state;

			hiprand_init((unsigned long long)clock(), i, 0, &state);

			tablero_d[tx + ty*altura] = hiprand_uniform(&state);
		}
	}
}

//Elimina las jewels recibidas, bajas las filas para rellenas, y genera arriba del todo jewels nuevas. TODO
void eliminarJewels(float* tablero, float* jewels_eliminadas,int dificultad, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	jewels_eliminadas = (float*)malloc(max * sizeof(float));

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	eliminarJewelsKernel <<<dimGrid, dimBlock >>>(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(jewels_eliminadas, jewels_eliminadas_d, size, hipMemcpyDeviceToHost);

	//Libera memoria
	hipFree(tablero_d);
	hipFree(jewels_eliminadas_d);
}

__global__ void analisisTableroKernel(float *tablero_d, float *jewels_eliminadas_d, int dificultad, int anchura, int altura) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	if (tablero_d[tx + altura*ty] == tablero_d[tx+1 + altura*ty] && tablero_d[tx + altura*ty] == tablero_d[tx-1 + altura*ty]) {
		jewels_eliminadas_d[0] = tx-1;
		jewels_eliminadas_d[1] = altura*ty;
		jewels_eliminadas_d[2] = tx;
		jewels_eliminadas_d[3] = altura*ty;
		jewels_eliminadas_d[4] = tx + 1;
		jewels_eliminadas_d[5] = altura*ty;
	}

	if (tablero_d[tx + altura*ty] == tablero_d[tx + altura*ty + 1] && tablero_d[tx + altura*ty] == tablero_d[tx + altura*ty - 1]) {
		jewels_eliminadas_d[0] = tx;
		jewels_eliminadas_d[1] = altura*ty-1;
		jewels_eliminadas_d[2] = tx;
		jewels_eliminadas_d[3] = altura*ty;
		jewels_eliminadas_d[4] = tx;
		jewels_eliminadas_d[5] = altura*ty+1;
	}
}

//CUDA CPU Function
void analisisTableroManual(int dificultad, float* tablero, int anchura, int altura) {
	float *tablero_d;
	float *jewels_eliminadas_d;
	int size = anchura * altura * sizeof(float);
	int max = 0;

	if (altura >= anchura) max = altura;
	else max = anchura;

	//Solo se eliminan 3 jewels, 2 coordenadas por jewel = 6 posiciones en el array
	float* jewels_eliminadas = (float*)malloc(max * sizeof(float));

	for (int i = 0; i < max; i++) {
		jewels_eliminadas[i] = -1;
	}

	//Tablero a GPU
	hipMalloc((void**)&tablero_d, size);
	hipMemcpy(tablero_d, tablero, size, hipMemcpyHostToDevice);

	//Jewels a eliminar a GPU
	hipMalloc((void**)&jewels_eliminadas_d, max * sizeof(float));
	hipMemcpy(jewels_eliminadas_d, jewels_eliminadas, max * sizeof(float), hipMemcpyHostToDevice);

	//Configuracion de ejecucion, 1 hilo por bloque, tantos bloques como celdas
	dim3 dimBlock(anchura, altura);
	dim3 dimGrid(1, 1);

	//Inicio del calculo, misma funcion de analisis en manual y automatico
	analisisTableroKernel <<<dimGrid, dimBlock>>>(tablero_d, jewels_eliminadas_d, dificultad, anchura, altura);

	//Transfiere las jewels a eliminar de la GPU al host
	hipMemcpy(jewels_eliminadas, jewels_eliminadas_d, size, hipMemcpyDeviceToHost);

	//Se eliminan las jewels seleccionadas, se bajan las superiores y se generan nuevas
	eliminarJewels(tablero, jewels_eliminadas, dificultad, anchura, altura);

	printTablero(tablero, anchura, altura);
	printf("Pulse una tecla para continuar...");
	getchar();
	/*if (jewels_eliminadas[0] == -1) {
		cudaFree(tablero_d);
		cudaFree(jewels_eliminadas_d);

		analisisTableroManual(dificultad, tablero, anchura, altura);
	}*/

}

//CUDA CPU Function.
void analisisTableroAutomatico() {

}

void intercambiarPosiciones(float* tablero, int jewel1_x, int jewel1_y, int direccion, int anchura, int altura) {
	int jewel2_x = jewel1_x;
	int jewel2_y = jewel1_y;
	switch (direccion) 
	{
		case 1: //Arriba
		{
			jewel2_y += 1;
			break;
		}
		case 2: //Abajo
		{
			jewel2_y -= 1;
			break;
		}
		case 3: //Izquierda
		{
			jewel2_x -= 1;
			break;
		}
		case 4: //Derecha
		{
			jewel2_x += 1;
			break;
		}
	}
	int aux1;

	aux1 = tablero[jewel2_x+jewel2_y*altura];

	tablero[jewel2_x+jewel2_y*altura] = tablero[jewel1_x+jewel1_y*altura];
	tablero[jewel1_x+jewel1_y*altura] = aux1;
}

int main() {
	//Matriz de tama�o variable de floats, un array de Altura*Anchura
	int anchura = 2;
	int altura = 2;
	int dificultad = 1;
	bool automatico = true;
	int TILE_WIDTH = 16;

	float *tablero;
	bool jugando = true;

	std::cout << "Anchura del tablero: ";
	std::cin >> anchura;

	std::cout << "Altura del tablero: ";
	std::cin >> altura;

	std::cout << "Elija dificultad: \n1.-Facil \n2.-Media \n3.-Dificil";
	std::cin >> dificultad;

	int seleccion;
	std::cout << "Automatico?   1.-SI   2.-NO";
	std::cin >> seleccion;

	switch (seleccion) {
	case 1: automatico = true; break;
	case 2: automatico = false; break;
	default: printf("Valor no valido.\n"); return -1;
	}

	tablero = (float*)malloc(altura * anchura * sizeof(float));

	//Se inicializa la matriz
	generacionInicialRandomJewels(tablero, dificultad, anchura, altura);

	//Bucle principal del juego
	while (jugando) {
		printTablero(tablero, anchura, altura);

		/*if (seleccion == 2)
			analisisTableroManual(dificultad, tablero, anchura, altura);
		else
			if (seleccion == 1)
				analisisTableroAutomatico();*/

		int jewel1_x = 0;
		int jewel1_y = 0;
		std::cout << "Posicion de la primera jewel a intercambiar (empiezan en 0)\n";
		std::cout << "X: ";
		std::cin >> jewel1_x;
		std::cout << "Y: ";
		std::cin >> jewel1_y;

		if (!((jewel1_x < anchura) && (jewel1_x >= 0) && (jewel1_y < altura) && (jewel1_y >= 0))) {
			printf("Posicion erronea.\n");
			continue;
		}

		int direccion = 0;
		std::cout << "Direccion a seguir para intercambio de posiciones: \n 1.-Arriba\n 2.-Abajo\n 3.-Izquierda\n 4.-Derecha";
		std::cin >> direccion;

		if (direccion > 4 && direccion > 1) {
			printf("Direccion erronea.\n");
			continue;
		}
		else {
			switch (direccion)
			{
			case 1: //Arriba
			{
				if (jewel1_y == altura)
				{
					printf("No se puede realizar el intercambio especificado.\n");
					continue;
				}
				break;
			}
			case 2: //Abajo
			{
				if (jewel1_y == 0)
				{
					printf("No se puede realizar el intercambio especificado.\n");
					continue;
				}
				break;
			}
			case 3: //Izquierda
			{
				if (jewel1_x == 0)
				{
					printf("No se puede realizar el intercambio especificado.\n");
					continue;
				}
				break;
			}
			case 4: //Derecha
			{
				if (jewel1_x == anchura - 1)
				{
					printf("No se puede realizar el intercambio especificado.\n");
					continue;
				}
				break;
			}
			}

			intercambiarPosiciones(tablero, jewel1_x, jewel1_y, direccion, anchura, altura);

		}

		/*const int width = 3;
		float A[width*width], B[width*width], C[width*width];
		for (int i = 0; i < (width*width); i++) {
			int valor = 0;
			std::cout << "Valor en A (de izquierda a derecha, por filas): ";
			std::cin >> valor;
			A[i] = valor;
			valor = 0;
			std::cout << "Valor en B (de izquierda a derecha, por filas): ";
			std::cin >> valor;
			B[i] = valor;
			C[i] = 0;
		}
		MatrixMultiplication(A, B, C, width);
		printf("Solucion: \n");
		for (int i = 0; i < (width*width); i++) {
			if (i%width == 0) { printf("\n"); }
			printf("%f ", C[i]);
		}

		int exit;
		scanf("%d", &exit);*/
	}
	return 0;
}